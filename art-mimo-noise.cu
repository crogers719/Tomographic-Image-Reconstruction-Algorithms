#include "hip/hip_runtime.h"
// HEADERS

#include <iostream>
#include <iomanip>
#include <string>
#include <limits>
#include <stdlib.h>
#include <fstream>
#include <math.h>
#include <time.h>

#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

#include "cuda_ptr.cuh"
using namespace std;

// DEFINITIONS

#define NX 192				//was 201
#define NY 192				//was 201
#define NT 401

#define NS 640 				//number of sensors

#define BLOCK_X 16
#define BLOCK_Y 16

#define HX 0.001f
#define HY 0.001f
#define H 0.001f

/* __constant__ float T = 1.3333e-04f; // 0.2f / 1500.f; */
#define DT 3.3333e-07f
/* __constant__ float fre = 125000.f; */
#define OMEGAC 7.8540e+05f
#define TAO 4.0000e-06f
#define TT 8.1573e-06f

// FUNCTIONS DECLARATION

void Ultrasonic_Tomography(int group_size, float target_epsilon, int max_iterations, int ti);

void IO_Files(
		host_ptr<float> const x, host_ptr<float> const y,
		host_ptr<float> const fo, host_ptr<float> const f)
{
	int i = 0, j = 0;

	// I/O Files

	ofstream x_file, y_file;
	ofstream fo_file;
	ofstream f_file;

	cerr << "writing x to 'dev_x.txt'...\n"
		 << "writing y to 'dev_y.txt'...\n"
		 << "writing f0 to 'dev_f0.txt'...\n"
		 << "writing f to 'dev_f.txt'...\n\n";

	x_file.open("dev_x.txt");
	y_file.open("dev_y.txt");
	fo_file.open("dev_f0.txt");
	f_file.open("dev_f.txt");

	for(i = 0; i < NX; i++) {
		x_file << x(i);
		x_file << "\n";
	}

	for(j = 0; j < NX; j++) {
		y_file << y(j);
		y_file << "\n";
	}

	for(j = 0; j < NY; j++) {
		for(i = 0; i < NX; i++) {
			fo_file << fo(i, j);
			fo_file << " ";
		}

		fo_file << "\n";
	}

	for(j = 0; j < NY; j++) {
		for(i = 0; i < NX; i++) {
			f_file << f(i, j);
			f_file << " ";
		}

		f_file << "\n";
	}

	x_file.close();
	y_file.close();
	fo_file.close();
	f_file.close();
}

float norm(host_ptr<float> A, int nx, int ny)
{
	float sum = 0;

	for (int j = 0; j < ny; ++j)
		for (int i = 0; i < nx; ++i)
			sum += A(i, j) * A(i, j);

	return sqrtf(sum);
}

void Position_Transducers(int *&ii, int *&jj, int num)
{
//returns the (x,y) coordinates of the number of total transducers
	int p = 0;
	ii = (int*)malloc(num * sizeof(int));
	jj = (int*)malloc(num * sizeof(int));


	for(p = 0; p < 160; p++)
	{
		ii[p] = 21 + (p + 1);
		jj[p] = 181;
	}

	for(p = 160; p < 320; p++)
	{
		ii[p] = 181;
		jj[p] = 181 - ((p + 1) - 160);
	}

	for(p = 320; p < 480; p++)
	{
		ii[p] = 181 - ((p + 1) - 320);
		jj[p] = 21;
	}

	for(p = 480; p < num; p++)
	{
		ii[p] = 21;
		jj[p] = 21 + ((p + 1) - 480);
	}
}

__global__ void field_setup(kernel_ptr<float> const x, kernel_ptr<float> const y, kernel_ptr<float> fo)
{
	// Map from threadIdx / BlockIdx to pixel position

	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;

	if((i < NX) && (j < NY))
	{
		float value = 0.f;

		float rc = 0.015f;
		float rp = 0.005f;

		float sc = 0.03f;
		float sp = 0.05f;

		if (powf(x(i), 2) + powf(y(j), 2) <= powf(rc, 2))
		{
			value = sc;
		}

		if (powf(x(i) - rc * cos(-30 * (3.14159265f / 180)), 2) + powf(y(j) - rc * sin(30 * (3.14159265f / 180)), 2) <= powf(rp, 2))
		{
			value = sp;
		}

		if (powf(x(i) + rc * cos(-30 * (3.14159265f / 180)), 2) + powf(y(j) - rc * sin(30 * (3.14159265f / 180)), 2) <= powf(rp, 2))
		{
			value = sp;
		}

		if (powf(x(i), 2) + powf(y(j) + rc, 2) <= powf(rp, 2))
		{
			value = sp;
		}

		fo(i, j) = value;
	}
}
__global__ void noise_setup (kernel_ptr<hiprandState> state)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int k = threadIdx.y + blockIdx.y * blockDim.y;
	int g = threadIdx.z + blockIdx.z * blockDim.z;
	
	int id = i* k*g;	

	hiprand_init (1234, id, 0, &state(id));
}

__global__ void propagation(
		int jp1, int jp2, int ip1, int ip2, 
		kernel_ptr<float> const f,
		kernel_ptr<float> u, 
		int k)
{
	// Map from threadIdx / BlockIdx to pixel position

	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;

	if(i < NX && j < NY) {
		float v = 1500.f * sqrtf(1.f + f(i, j));
		float r = v * DT / HX;
		float s = 2.f - 4.f * r * r;

		float val; // will hold new u at (i, j, k + 1)

		// not at boundary
		if (i != 0 && i != NX - 1 && j != 0 && j != NY - 1) {

			val =
				r * r *
				(u(i+1, j, k) +
				 u(i-1, j, k) +
				 u(i, j-1, k) +
				 u(i, j+1, k)) +
				s * u(i, j, k) -
				u(i, j, k-1);

			// at sensor, k <= 24
			if (j + 1 >= jp1 && j + 1 <= jp2 && i + 1 >= ip1 && i + 1 <= ip2 && k + 1 <= 24) {
				float t = k * DT - TT;

				// add wave value
				val +=
					v * v * DT * DT *
					cosf(OMEGAC * t) *
					expf(-(t * t) / (2.f * TAO * TAO));
			}
		}

		// at boundary
		else {

			// boundary booleans
			bool top = (j == 0);
			bool bottom = (j == NY - 1);
			bool left = (i == 0);
			bool right = (i == NX - 1);

			// index variables for different boundary cases
			int ja = top ? (j + 1) : bottom ? (j - 1) : j;
			int jb = top ? (j + 2) : bottom ? (j - 2) : j;

			int ia = left ? (i + 1) : right ? (i - 1) : i;
			int ib = left ? (i + 2) : right ? (i - 2) : i;

			val =
				(2.f - 2.f * r - r * r) * u(i, j, k) +
				2.f * r * (1.f + r) * u(ia, ja, k) -
				r * r * u(ib, jb, k) +
				(2.f * r - 1.f) * u(i, j, k-1) -
				2.f * r * u(ia, ja, k-1);
		}

		u(i, j, k+1) = val;

		/* if (k+1 == NT - 1) */
			/* printf("%e \t", val); */
	}
}

__global__ void propagation_at_corners(kernel_ptr<float> u)
{
	int k = threadIdx.x + blockIdx.x * blockDim.x;

	if (k < NT) {
		u(0, 0, k) =
			1.f / 2.f * (u(0, 1, k) + u(1, 0, k));

		u(NX-1, 0, k) =
			1.f / 2.f * (u(NX-2, 0, k) + u(NX-1, 1, k));

		u(0, NY-1, k) =
			1.f / 2.f * (u(0, NY-2, k) + u(1, NY-1, k));

		u(NX-1, NY-1, k) =
			1.f / 2.f * (u(NX-2, NY-1, k) + u(NX-1, NY-2, k));
	}
}

__global__ void initial_signal(
		kernel_ptr<float> const u,
		kernel_ptr<float> g_bottom,
		kernel_ptr<float> g_right,
		kernel_ptr<float> g_top,
		kernel_ptr<float> g_left,
		kernel_ptr <hiprandState> state,
		int p)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int k = threadIdx.y + blockIdx.y * blockDim.y;
	
	int id = i*k;
	float value, noise;

	hiprandState localState =state(id);
	value = hiprand_uniform (&localState);

	noise = 0.9f +value * (1.1f -0.9f);

	if (i > 20 && i < 180 && k > 1 && k < NT) {
		// store values at bottom sensor row of u
		g_bottom(i, k, p) =
			u(i, 180, k) *noise;

		// store values at top sensor row of u
		g_top(i, k, p) =
			u(i, 20, k) *noise;

		// store values at right sensor column of u
		g_right(i, k, p) =
			u(180, i, k) *noise;

		// store values at left sensor column of u
		g_left(i, k, p) =
			u(20, i, k) *noise;
	}
}

__global__ void difference_signal(
		kernel_ptr<float> const u,
		kernel_ptr<float> const g_bottom,
		kernel_ptr<float> const g_right,
		kernel_ptr<float> const g_top,
		kernel_ptr<float> const g_left,
		kernel_ptr<float> rr_bottom,
		kernel_ptr<float> rr_right,
		kernel_ptr<float> rr_top,
		kernel_ptr<float> rr_left,
		kernel_ptr <hiprandState> state,		
		int p)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int k = threadIdx.y + blockIdx.y * blockDim.y;
	
	int id = i*k;
	float  value, noise;

	hiprandState localState =state (id);
	value=hiprand_uniform (&localState);

	noise= 0.9f + value * (1.1f - 0.9f);

	if (i > 20 && i < 180 && k > 1 && k < NT) {
		// store difference at time k of original signal
		// and current signal at bottom sensor row
		rr_bottom(i, k) =
			g_bottom(i, k, p) -
			u(i, 180, k)*noise;

		/* printf("%e ", rr_bottom(i+21, k+2)); */

		// store difference at time k of original signal
		// and current signal at top sensor row
		rr_top(i, k) =
			g_top(i, k, p) -
			u(i, 20, k) *noise;

		// store difference at time k of original signal
		// and current signal at right sensor column
		rr_right(i, k) =
			g_right(i, k, p) -
			u(180, i, k) *noise;

		// store difference at time k of original signal
		// and current signal at left sensor column
		rr_left(i, k) =
			g_left(i, k, p) -
			u(20, i, k) *noise;
	}
}

__global__ void backpropagation1(
		kernel_ptr<float> z,
		kernel_ptr<float> const f,
		int k)
{
	// Map from threadIdx / BlockIdx to pixel position

	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;

	if(i >= 1 && i < (NX - 1) && j >= 1 && j < (NY - 1))
	{
		z(i, j, k) =
			1500.f * 1500.f * (DT * DT) *
			((1.f + f(i, j-1)) * z(i, j-1, k+1) +
			 (1.f + f(i, j+1)) * z(i, j+1, k+1) +
			 (1.f + f(i-1, j)) * z(i-1, j, k+1) +
			 (1.f + f(i+1, j)) * z(i+1, j, k+1) -
			 4.f * (1.f + f(i, j)) *
			 z(i, j, k+1)) / (H * H) +
			2.f * z(i, j, k+1) -
			z(i, j, k+2);

		/* if (k == 1) */
			/* printf("%e \t", z(i, j, k)); */
	}
}

__global__ void backpropagation2(
		kernel_ptr<float> z,
		kernel_ptr<float> const rr_bottom,
		kernel_ptr<float> const rr_right,
		kernel_ptr<float> const rr_top,
		kernel_ptr<float> const rr_left,
		int k)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;

	if(i >= 21 && i < 180) {
		z(i, 180, k) =
			z(i, 179, k) +
			rr_bottom(i, k) * H * 1000.f;

		z(i, 20, k) =
			z(i, 21, k) +
			rr_top(i, k) * H * 1000.f;

		z(180, i, k) =
			z(179, i, k) +
			rr_right(i, k) * H * 1000.f;

		z(20, i, k) =
			z(21, i, k) +
			rr_left(i, k) * H * 1000.f;
	}

	if (i >= 1 && i < (NX - 1)) {
		z(i, 0, k) =
			z(i, 1, k);

		z(i, NY-1, k) =
			z(i, NY-2, k);

		z(0, i, k) =
			z(1, i, k);

		z(NX-1, i, k) =
			z(NX-2, i, k);
	}

	else if (i == 0) {
		z(0, 0, k) =
			(z(1, 0, k) +
			 z(0, 1, k)) / 2.f;

		z(NX-1, 0, k) =
			(z(NX-2, 0, k) +
			 z(NX-1, 1, k)) / 2.f;

		z(0, NY-1, k) =
			(z(1, NY-1, k) +
			 z(0, NY-2, k)) / 2.f;

		z(NX-1, NY-1, k) =
			(z(NX-2, NY-1, k) +
			 z(NX-1, NY-2, k)) / 2.f;
	}
}

__global__ void laplace(
		kernel_ptr<float> const u,
		kernel_ptr<float> Lu)
{
	// Map from threadIdx / BlockIdx to pixel position

	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;
	int k = threadIdx.z + blockIdx.z * blockDim.z;

	if (i < NX && j < NY && (k + 1) < NT) {

		int j_prev = (j > 0) ? j - 1 : j;
		int j_next = (j < NY - 1) ? j + 1 : j;

		int i_prev = (i > 0) ? i - 1 : i;
		int i_next = (i < NX - 1) ? i + 1 : i;

		Lu(i, j, k+1) =
			(u(i, j_prev, k+1) +
			 u(i, j_next, k+1) +
			 u(i_prev, j, k+1) +
			 u(i_next, j, k+1) -
			 4.f * u(i, j, k+1)) / (H * H);
	}
}

__global__ void laplace_corners(kernel_ptr<float> const u, kernel_ptr<float> Lu)
{
	int k = threadIdx.x + blockIdx.x * blockDim.x;

	if ((k + 1) < NT) {
		Lu(0, 0, k+1) =
			(Lu(1, 0, k+1) +
			 Lu(0, 1, k+1)) / 2.f;

		Lu(NX-1, 0, k+1) =
			(Lu(NX-2, 0, k+1) +
			 Lu(NX-1, 1, k+1)) / 2.f;

		Lu(0, NY-1, k+1) =
			(Lu(1, NY-1, k+1) +
			 Lu(0, NY-2, k+1)) / 2.f;

		Lu(NX-1, NY-1, k+1) =
			(Lu(NX-2, NY-1, k+1) +
			 Lu(NX-1, NY-2, k+1)) / 2.f;
	}
}

__global__ void update_differential(
		kernel_ptr<float> df,
		kernel_ptr<float> const z,
		kernel_ptr<float> const Lu,
		kernel_ptr<float> const f)
{
	// Map from threadIdx / BlockIdx to pixel position

	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;
	int k = threadIdx.z + blockIdx.z * blockDim.z;

	if(i < NX && j < NY && (k + 1) < NT) {

		atomicAdd(
			&df(i, j),
			z(i, j, k+1) *
			Lu(i, j, k+1) /
			(1.f + f(i, j)));
	}
}

__global__ void update_field(
		kernel_ptr<float> f,
		kernel_ptr<float> const df,
		kernel_ptr<float> f_minus_fo,
		kernel_ptr<float> const fo)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;

	if (i < NX && j < NY)
	{
		bool in_sensor_field = (i >= 21) && (i < 180) && (j >= 21) && (j < 180);

		float alpha = in_sensor_field ? 1.f : 0.f;

		f(i, j) += 20000.f * alpha * df(i, j);
		f_minus_fo(i, j) = f(i, j) - fo(i, j);
	}
}


// MAIN PROGRAM

int main(int argc, char **argv)
{
	if (argc != 4) {
		cerr << "Usage: " << argv[0] << " <sensor group size> <target epsilon> <max iterations>\n\n";
		exit(1);
	}

	int group_size = stoi(argv[1]);
	float target_epsilon = stof(argv[2]);
	int max_iterations = stoi(argv[3]);

	if (max_iterations == -1)
		max_iterations = numeric_limits<int>::max();

	// Time measuring variables

	int ti = 0, tf = 0;

	// Function Execution

	// set floting-point precision on stdout and stderr
	cout << fixed << setprecision(10);
	cerr << fixed << setprecision(10);

	cerr << "Ultrasonic Tomography Running:\n\n";

	ti = clock();
	cerr << "ti = " << ti << "\n";

	Ultrasonic_Tomography(group_size, target_epsilon, max_iterations, ti);
	hipDeviceReset();

	tf = clock();
	cerr << "tf = " << tf << "\n"
		 << "tt = " << tf - ti << "\n"
		 << "Total Seconds = " << (float)(tf - ti) / CLOCKS_PER_SEC << "\n";

	// End of the program

	return 0;
}

inline int grid_size(int n, int threads)
{
	return ceil(float(n) / threads);
}

// FUNCTIONS DEFINITION

void Ultrasonic_Tomography(int group_size, float target_epsilon, int max_iterations, int ti)
{
	// Simulation Variables

	int i = 0, j = 0, k = 0;

	host_ptr<float> x(NX);
	host_ptr<float> y(NY);
	host_ptr<float> fo(NX, NY);

	dim3 Block_Size(BLOCK_X, BLOCK_Y);
	dim3 Grid_Size(grid_size(NX, BLOCK_X), grid_size(NY, BLOCK_Y));

	device_ptr<float> dev_x(NX);
	device_ptr<float> dev_y(NY);
	device_ptr<float> dev_fo(NX, NY);

	device_ptr<float> dev_u(NX, NY, NT);

	dev_u.set(0.f);

	device_ptr<float> dev_g_bottom(NX, NT, NS / group_size);
	device_ptr<float> dev_g_right(NX, NT, NS / group_size);
	device_ptr<float> dev_g_top(NX, NT, NS / group_size);
	device_ptr<float> dev_g_left(NX, NT, NS / group_size);

	dev_g_bottom.set(0.f);
	dev_g_right.set(0.f);
	dev_g_top.set(0.f);
	dev_g_left.set(0.f);

	// Environment Initialization

	for(i = 0; i < NX; i++)
	{
		x(i) = -0.1f + i * HX;
	}

	for(j = 0; j < NY; j++)
	{
		y(j) = -0.1f + j * HY;
	}

	copy(dev_x, x);
	copy(dev_y, y);

	field_setup<<<Grid_Size, Block_Size>>>(dev_x, dev_y, dev_fo);

	copy(fo, dev_fo);

	//initialize state for generating random noise
	host_ptr<hiprandState> state (NX, NY);
	device_ptr<hiprandState> dev_state (NX, NY);
	
	copy (state, dev_state);

	dim3 threads_noise (NX, 1);
	dim3 grid_noise(
			grid_size(NX, threads_noise.x),
			grid_size (NY, threads_noise.y));

	noise_setup<<<grid_noise, threads_noise>>>(dev_state);


	// Position of the transducers
	int *ii, *jj;
	Position_Transducers(ii, jj, NS);


	dim3 threads_propagation(NX, 1, 1);
	dim3 grid_propagation(
			grid_size(NX, threads_propagation.x),
			grid_size(NY, threads_propagation.y));

	dim3 threads_signal(NX, 1);
	dim3 grid_signal(
			grid_size(NX, threads_signal.x),
			grid_size(NT, threads_signal.y));

	int p;
	for(p = 0; p < NS; p += group_size)
	{
		dev_u.set(0.f);

		int jp1 = jj[p];
		int jp2 = jj[p + group_size - 1];
		int ip1 = ii[p];
		int ip2 = ii[p + group_size - 1];

		if (jp2 < jp1)
		{
			int jp = jp1;
			jp1 = jp2;
			jp2 = jp;
		}

		if (ip2 < ip1)
		{
			int ip = ip1;
			ip1 = ip2;
			ip2 = ip;
		}

		// Boundary

		for(k = 1; k < NT - 1; k++)
		{
			propagation<<<grid_propagation, threads_propagation>>>(jp1, jp2, ip1, ip2, dev_fo, dev_u, k);
		}

		// Four corners

		propagation_at_corners<<<NT, 1>>>(dev_u);

		initial_signal<<<grid_signal, threads_signal>>>(dev_u, dev_g_bottom, dev_g_right, dev_g_top, dev_g_left, dev_state, p / group_size);
	}

	// Kaczmarz method
	// propagation

	device_ptr<float> dev_rr_bottom(NX, NT);
	device_ptr<float> dev_rr_right(NX, NT);
	device_ptr<float> dev_rr_top(NX, NT);
	device_ptr<float> dev_rr_left(NX, NT);

	dev_rr_bottom.set(0.f);
	dev_rr_right.set(0.f);
	dev_rr_top.set(0.f);
	dev_rr_left.set(0.f);

	device_ptr<float> dev_z(NX, NY, NT+1);
	device_ptr<float> dev_Lu(NX, NY, NT);
	dev_Lu.set(0.f);

	device_ptr<float> dev_f(NX, NY);
	dev_f.set(0.f);

	device_ptr<float> dev_df(NX, NY);
	device_ptr<float> dev_f_minus_fo(NX, NY);

	// Allocation


	host_ptr<float> f(NX, NY);
	host_ptr<float> f_minus_fo(NX, NY);

	// initialize epsilon values
	float prev_epsilon = std::numeric_limits<float>::infinity();
	float curr_epsilon = -std::numeric_limits<float>::infinity();
	float file_epsilon = std::numeric_limits<float>::infinity();

	cerr << "writing convergence to 'art_convergence.txt'...\n"
		 << "writing time to 'art_time.txt'...\n";

	ofstream convergence_file("art_convergence.txt");
	ofstream time_file("art_time.txt");

	dim3 threads_diff_signal(NX, 1);
	dim3 grid_diff_signal(
			grid_size(NX, threads_diff_signal.x),
			grid_size(NT, threads_diff_signal.y));

	dim3 threads_backpropagation1(NX, 1, 1);
	dim3 grid_backpropagation1(
			grid_size(NX, threads_backpropagation1.x),
			grid_size(NY, threads_backpropagation1.y));

	dim3 threads_laplace(96, 2, 1);
	dim3 grid_laplace(
			grid_size(NX, threads_laplace.x),
			grid_size(NY, threads_laplace.y),
			grid_size(NT, threads_laplace.z));

	dim3 threads_differential(96, 2, 1);
	dim3 grid_differential(
			grid_size(NX, threads_differential.x),
			grid_size(NY, threads_differential.y),
			grid_size(NT, threads_differential.z));

	for(int iter = 0; iter < max_iterations; iter++)
	{
		cout << "\nIter: " << iter << "\n";
		dev_u.set(0.f);

		for(p = 0; p < NS; p += group_size)
		{
			int jp1 = jj[p];
			int jp2 = jj[p + group_size - 1];
			int ip1 = ii[p];
			int ip2 = ii[p + group_size - 1];

			if (jp2 < jp1)
			{
				int jp = jp1;
				jp1 = jp2;
				jp2 = jp;
			}

			if (ip2 < ip1)
			{
				int ip = ip1;
				ip1 = ip2;
				ip2 = ip;
			}

			// Boundary

			for(k = 1; k < NT - 1; k++)
			{
				propagation<<<grid_propagation, threads_propagation>>>(jp1, jp2, ip1, ip2, dev_f, dev_u, k);
			}

			// Four corners

			propagation_at_corners<<<NT, 1>>>(dev_u);
			difference_signal<<<grid_diff_signal, threads_diff_signal>>>(dev_u, dev_g_bottom, dev_g_right, dev_g_top, dev_g_left, dev_rr_bottom, dev_rr_right, dev_rr_top, dev_rr_left, dev_state, p / group_size);

			dev_z.set(0.f);

			for(k = NT - 2; k > 0; k--)
			{
				backpropagation1<<<grid_backpropagation1, threads_backpropagation1>>>(dev_z, dev_f, k);
				backpropagation2<<<NX, 1>>>(dev_z, dev_rr_bottom, dev_rr_right, dev_rr_top, dev_rr_left, k);
			}

			laplace<<<grid_laplace, threads_laplace>>>(dev_u, dev_Lu);
			laplace_corners<<<NT, 1>>>(dev_u, dev_Lu);

			dev_df.set(0.f);
			update_differential<<<grid_differential, threads_differential>>>(dev_df, dev_z, dev_Lu, dev_f);

			update_field<<<Grid_Size, Block_Size>>>(dev_f, dev_df, dev_f_minus_fo, dev_fo);
		}

		copy(f_minus_fo, dev_f_minus_fo);

		curr_epsilon = norm(f_minus_fo, NX, NY) / norm(fo, NX, NY) * 100.f;
		float current_t = (float)(clock()-ti) / CLOCKS_PER_SEC;

		if (file_epsilon - curr_epsilon > 0.2f) {
			convergence_file << curr_epsilon << " ";
			time_file << current_t << " ";
			file_epsilon = curr_epsilon;
		}

		cout << "epsilon = " << curr_epsilon << "\n";

		// stop if reached target epsilon
		if (curr_epsilon <= target_epsilon) {
			break;
		}

		// stop if epsilon diverges
		if (curr_epsilon > prev_epsilon ||
				std::isnan(curr_epsilon)) {
			break;
		}

		// update prev_epsilon
		prev_epsilon = curr_epsilon;
	}

	cout << endl;

	copy(f, dev_f);

	IO_Files(x, y, fo, f);

	// Free Variables

	delete [] ii;
	delete [] jj;

}
