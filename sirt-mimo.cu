#include "hip/hip_runtime.h"
/**********HEADERS**********/

#include <iostream>
#include <iomanip>
#include <string>
#include <limits>
#include <stdlib.h>
#include <fstream>
#include <math.h>
#include <time.h>

#include "cuda_ptr.cuh"
using namespace std;

/**********DEFINING CONSTANTS***********/

#define NX 192				//was 201
#define NY 192				//was 201
#define NT 401

#define NS 640 				//number of sensors

#define BLOCK_X 16
#define BLOCK_Y 16

#define HX 0.001f
#define HY 0.001f
#define H 0.001f

/* __constant__ float T = 1.3333e-04f; // 0.2f / 1500.f; */
#define DT 3.3333e-07f
/* __constant__ float fre = 125000.f; */
#define OMEGAC 7.8540e+05f
#define TAO 4.0000e-06f
#define TT 8.1573e-06f

/**********FUNCTION DECLARATION**********/

//Host Functions
void Ultrasonic_Tomography(int, float, int, int, float);
void Position_Transducers(host_ptr<int>, host_ptr<int>, int);
void IO_Files(host_ptr<float> const, host_ptr<float> const);
float norm(host_ptr<float>, int, int);

//In-Line Functions
inline int grid_size(int, int);
template <typename T> __host__ __device__ void minmax(T &a, T &b);

//Device Functions
__global__ void field_setup(kernel_ptr<float> const, kernel_ptr<float> const, kernel_ptr<float>);
__global__ void propagation(kernel_ptr<int> const, kernel_ptr<int> const,	kernel_ptr<float> const, kernel_ptr<float>,	int, int, int);
__global__ void propagation_at_corners(kernel_ptr<float>,	int);
__global__ void initial_signal(kernel_ptr<float> const,	kernel_ptr<float>, kernel_ptr<float>,	kernel_ptr<float>,kernel_ptr<float>,int);
__global__ void difference_signal(kernel_ptr<float> const,kernel_ptr<float> const, kernel_ptr<float> const, kernel_ptr<float> const, kernel_ptr<float> const,	kernel_ptr<float>,kernel_ptr<float>,	kernel_ptr<float>, kernel_ptr<float>,	int);
__global__ void backpropagation1(kernel_ptr<float>, kernel_ptr<float> const,int, int);
__global__ void backpropagation2(kernel_ptr<float>,	kernel_ptr<float> const, kernel_ptr<float> const,	kernel_ptr<float> const,kernel_ptr<float> const, int, int);
__global__ void laplace(kernel_ptr<float> cons,	kernel_ptr<float>, int);
__global__ void laplace_corners(kernel_ptr<float> const, kernel_ptr<float>,	int);
__global__ void update_differential(kernel_ptr<float>,kernel_ptr<float>,kernel_ptr<float> const, kernel_ptr<float> const ,int);
__global__ void update_field(kernel_ptr<float> f, kernel_ptr<float> const df, kernel_ptr<float> f_minus_fo, kernel_ptr<float> const fo,	float omega);



/***************MAIN PROGRAM***************/

int main(int argc, char **argv)
{
	//Command Line Argument Processing
	if (argc != 5) {
		cerr << "Usage: " << argv[0] << " <sensor group size> <target epsilon> <max iterations> <omega>\n\n";
		exit(1);
	}

	int group_size = stoi(argv[1]);
	float target_epsilon = stof(argv[2]);
	int max_iterations = stoi(argv[3]);
	float omega = stof(argv[4]);

	if (max_iterations == -1)
		max_iterations = numeric_limits<int>::max();

	// Time Measuring Variables
	int ti = 0, tf = 0;

	// set floating-point precision on stdout and stderr
	cout << fixed << setprecision(10);
	cerr << fixed << setprecision(10);

	cerr << "Ultrasonic Tomography Running:\n\n";

	//Initial time
	ti = clock();
	cerr << "ti = " << ti << "\n";

	Ultrasonic_Tomography(group_size, target_epsilon, max_iterations, ti, omega);
	hipDeviceReset();

	//Calculate total time
	tf = clock();
	cerr << "tf = " << tf << "\n"
		 << "tt = " << tf - ti << "\n"
		 << "Total Seconds = " << (float)(tf - ti) / CLOCKS_PER_SEC << "\n";
}



/**********HOST FUNCTION DEFINITIONS**********/

void Ultrasonic_Tomography(int group_size, float target_epsilon, int max_iterations, int ti, float omega)
{
	// environment initialization

	host_ptr<float> x(NX);
	host_ptr<float> y(NY);
	device_ptr<float> dev_x(NX);
	device_ptr<float> dev_y(NY);

	for (int i = 0; i < NX; i++)
		x(i) = -0.1f + i * HX;

	for (int j = 0; j < NY; j++)
		y(j) = -0.1f + j * HY;

	copy(dev_x, x);
	copy(dev_y, y);

	// fo(i, j) =
	//    ground truth value at pos (i, j) of field
	host_ptr<float> fo(NX, NY);
	device_ptr<float> dev_fo(NX, NY);

	// kernel launch parameters for field kernels
	dim3 threads_field(NX, 1);
	dim3 grid_field(
		grid_size(NX, threads_field.x),
		grid_size(NY, threads_field.y));

	// initialize the ground truth field
	field_setup<<<grid_field, threads_field>>>(dev_x, dev_y, dev_fo);

	// copy from device to host
	copy(fo, dev_fo);

	// Position of the transducers
	host_ptr<int> ii(NS);
	host_ptr<int> jj(NS);
	device_ptr<int> dev_ii(NS);
	device_ptr<int> dev_jj(NS);

	Position_Transducers(ii, jj, NS);

	// copy from host to device
	copy(dev_ii, ii);
	copy(dev_jj, jj);

	// Ng = number of sensor groups that will be launched in parallel
	int Ng = NS / group_size;

	// u(i, j, k, g) =
	//    wave propagation at pos (i, j) of field, at time k, from sensor group g
	device_ptr<float> dev_u(NX, NY, NT, Ng);
	dev_u.set(0.f);

	// kernel launch parameters for propagation
	dim3 threads_propagation(NX, 1, 1);
	dim3 grid_propagation(
			grid_size(NX, threads_propagation.x),
			grid_size(NY, threads_propagation.y),
			grid_size(Ng, threads_propagation.z));

	// kernel launch parameters for propagation_at_corners
	dim3 threads_prop_corners(NT, 1);
	dim3 grid_prop_corners(
			grid_size(NT, threads_prop_corners.x),
			grid_size(Ng, threads_prop_corners.y));

	// initial wave propagation over fo
	for (int k = 1; k < NT - 1; ++k)
		propagation<<<grid_propagation, threads_propagation>>>(dev_ii, dev_jj, dev_fo, dev_u, k, group_size, Ng);

	propagation_at_corners<<<grid_prop_corners, threads_prop_corners>>>(dev_u, Ng);

	// gg_xxx(i, k, g) =
	//    initial signal at pos i in row/column xxx
	//    at time k, from sensor group
	//    e.g g_bottom stores the bottom row,
	//        g_right stores the right column
	device_ptr<float> dev_g_bottom(NX, NT, Ng);
	device_ptr<float> dev_g_right(NY, NT, Ng);
	device_ptr<float> dev_g_top(NX, NT, Ng);
	device_ptr<float> dev_g_left(NY, NT, Ng);

	dev_g_bottom.set(0.f);
	dev_g_right.set(0.f);
	dev_g_top.set(0.f);
	dev_g_left.set(0.f);

	// kernel launch parameters for initial_signal
	dim3 threads_signal(NX, 1, 1);
	dim3 grid_signal(
			grid_size(NX, threads_signal.x),
			grid_size(NT, threads_signal.y),
			grid_size(Ng, threads_signal.z));

	// store initial signal of wave at sensor positions of u in g
	initial_signal<<<grid_signal, threads_signal>>>(dev_u, dev_g_bottom, dev_g_right, dev_g_top, dev_g_left, Ng);

	// Kaczmarz method
	// propagation

	// rr_xxx(i, k, g) =
	//    difference signal between xxx sensors in u and gg_xxx
	//    at time k, from sensor group g
	device_ptr<float> dev_rr_bottom(NX, NT, Ng);
	device_ptr<float> dev_rr_right(NX, NT, Ng);
	device_ptr<float> dev_rr_top(NX, NT, Ng);
	device_ptr<float> dev_rr_left(NX, NT, Ng);

	dev_rr_bottom.set(0.f);
	dev_rr_right.set(0.f);
	dev_rr_top.set(0.f);
	dev_rr_left.set(0.f);

	// z(i, j, k, g) =
	//    wave back propagation at pos (i, j) of field,
	//    at time k, from sensor group g
	device_ptr<float> dev_z(NX, NY, NT+1, Ng);

	// Lu(i, j, k, g) =
	//    result of applying the Laplace operator to u(i, j, k, g)
	device_ptr<float> dev_Lu(NX, NY, NT, Ng);
	dev_Lu.set(0.f);

	// f(i, j) =
	//    current reconstruction of field at pos (i, j)
	host_ptr<float> f(NX, NY);
	device_ptr<float> dev_f(NX, NY);
	dev_f.set(0.f);

	// df(i, j) =
	//    discretized differential of f(i, j)
	device_ptr<float> dev_df(NX, NY);

	// f_minus_fo(i, j)
	//    difference of field and ground truth at pos (i, j)
	host_ptr<float> f_minus_fo(NX, NY);
	device_ptr<float> dev_f_minus_fo(NX, NY);

	// initialize epsilon values
	float prev_epsilon = std::numeric_limits<float>::infinity();
	float curr_epsilon = -std::numeric_limits<float>::infinity();
	float file_epsilon = std::numeric_limits<float>::infinity();

	cerr << "writing convergence to 'sirt_convergence.txt'...\n"
		 << "writing time to 'sirt_time.txt'...\n";

	ofstream convergence_file("sirt_convergence.txt");
	ofstream time_file("sirt_time.txt");

	// kernel launch parameters for difference_signal
	dim3 threads_diff_signal(NX, 1, 1);
	dim3 grid_diff_signal(
			grid_size(NX, threads_diff_signal.x),
			grid_size(NT, threads_diff_signal.y),
			grid_size(Ng, threads_diff_signal.z));

	// kernel launch parameters for backpropagation1
	dim3 threads_backpropagation1(NX, 1, 1);
	dim3 grid_backpropagation1(
			grid_size(NX, threads_backpropagation1.x),
			grid_size(NY, threads_backpropagation1.y),
			grid_size(Ng, threads_backpropagation1.z));

	// kernel launch parameters for backpropagation2
	dim3 threads_backpropagation2(Ng, 1);
	dim3 grid_backpropagation2(
			grid_size(NX, threads_backpropagation2.x),
			grid_size(Ng, threads_backpropagation2.y));

	// kernel launch parameters for laplace
	dim3 threads_laplace(NX, 1, 1);
	dim3 grid_laplace(
			grid_size(NX * NY, threads_laplace.x),
			grid_size(NT, threads_laplace.y),
			grid_size(Ng, threads_laplace.z));

	// kernel launch parameters for laplace_corners
	dim3 threads_laplace_corners(NX, 1, 1);
	dim3 grid_laplace_corners(
			grid_size(NX * NY, threads_laplace.x),
			grid_size(NT, threads_laplace.y),
			grid_size(Ng, threads_laplace.z));

	// kernel launch parameters for update_differential
	dim3 threads_differential(NX, 1, 1);
	dim3 grid_differential(
			grid_size(NX * NY, threads_differential.x),
			grid_size(NT, threads_differential.y),
			grid_size(Ng, threads_differential.z));

	cerr << "group size:     " << group_size << "\n"
		 << "target epsilon: " << setprecision(2) << target_epsilon << "\n"
		 << "omega:          " << setprecision(0) << omega << "\n\n";

	for(int iter = 0; iter < max_iterations; iter++)
	{
		cout << "\nIter: " << iter << "\n";
	
		dev_u.set(0.f);
		dev_z.set(0.f);
		dev_df.set(0.f);

		// propagate wave over field, store in u
		for (int k = 1; k < NT - 1; ++k)
			propagation<<<grid_propagation, threads_propagation>>>(dev_ii, dev_jj, dev_f, dev_u, k, group_size, Ng);

		propagation_at_corners<<<grid_prop_corners, threads_prop_corners>>>(dev_u, Ng);

		// store difference signal of u at sensor positions and initial signal at g in rr
		difference_signal<<<grid_diff_signal, threads_diff_signal>>>(dev_u, dev_g_bottom, dev_g_right, dev_g_top, dev_g_left, dev_rr_bottom, dev_rr_right, dev_rr_top, dev_rr_left, Ng);

		// do back propagation of wave over field, store in z
		for(int k = NT - 2; k > 0; k--)
		{
			backpropagation1<<<grid_backpropagation1, threads_backpropagation1>>>(dev_z, dev_f, k, Ng);
			backpropagation2<<<grid_backpropagation2, threads_backpropagation2>>>(dev_z, dev_rr_bottom, dev_rr_right, dev_rr_top, dev_rr_left, k, Ng);
		}

		// apply Laplace operator to u, store in Lu
		laplace<<<grid_laplace, threads_laplace>>>(dev_u, dev_Lu, Ng);
		laplace_corners<<<grid_laplace_corners, threads_laplace_corners>>>(dev_u, dev_Lu, Ng);

		// update differential of f, store in df
		update_differential<<<grid_differential, threads_differential>>>(dev_df, dev_z, dev_Lu, dev_f, Ng);

		// update f and f_minus_fo
		update_field<<<grid_field, threads_field>>>(dev_f, dev_df, dev_f_minus_fo, dev_fo, omega);

		// error calculation

		// copy from device to host
		copy(f_minus_fo, dev_f_minus_fo);

		curr_epsilon = norm(f_minus_fo, NX, NY) / norm(fo, NX, NY) * 100.f;
		float current_t = (float)(clock()-ti) / CLOCKS_PER_SEC;

		if (file_epsilon - curr_epsilon > 0.2f) {
			convergence_file << curr_epsilon << " ";
			time_file << current_t << " ";
			file_epsilon = curr_epsilon;
		}

		cout << "epsilon = " << setprecision(8) << curr_epsilon << "\n";

		// stop if reached target epsilon
		if (curr_epsilon <= target_epsilon) {
			cerr << "reached target epsilon = " << setprecision(2) << target_epsilon << ", at iter = " << iter << ", epsilon = " << setprecision(8) << curr_epsilon << "\n\n";
			break;
		}

		// stop if epsilon diverges
		if (curr_epsilon > prev_epsilon ||
				std::isnan(curr_epsilon)) {
			cerr << "diverged at iter = " << iter << ", epsilon = " << setprecision(8) << curr_epsilon << "\n\n";
			break;
		}

		// update prev_epsilon
		prev_epsilon = curr_epsilon;
	}

	cout << endl;

	// copy from device to host
	copy(f, dev_f);

	IO_Files(fo, f);

	size_t free, total;
	hipMemGetInfo(&free, &total);

	cerr << fixed << setprecision(4);

	cerr << "used mem:  " << float(total - free) / (1024 * 1024) << " MB\n"
		 << "free mem:  " << float(free) / (1024 * 1024)  << " MB\n"
		 << "total mem: " << float(total) / (1024 * 1024) << " MB\n\n";
}

void IO_Files(host_ptr<float> const fo, host_ptr<float> const f)
{
	int i = 0, j = 0;

	// IO Files
	ofstream fo_file;
	ofstream f_file;

	cerr << "writing f0 to 'dev_f0.txt'...\n"
		 << "writing f to 'dev_f.txt'...\n\n";

	//ground truth file
	fo_file.open("dev_f0.txt");
	//reconstructed image file
	f_file.open("dev_f.txt");

	for(j = 0; j < NY; j++) {
		for(i = 0; i < NX; i++) {
			fo_file << fo(i, j);
			fo_file << " ";
		}

		fo_file << "\n";
	}

	for(j = 0; j < NY; j++) {
		for(i = 0; i < NX; i++) {
			f_file << f(i, j);
			f_file << " ";
		}

		f_file << "\n";
	}

	fo_file.close();
	f_file.close();
}

float norm(host_ptr<float> A, int nx, int ny)
{
	float sum = 0;

	for (int j = 0; j < ny; ++j)
		for (int i = 0; i < nx; ++i)
			sum += A(i, j) * A(i, j);

	return sqrtf(sum);
}

void Position_Transducers(host_ptr<int> ii, host_ptr<int> jj, int num)
{
	//returns the (x,y) coordinates of the number of total transducers
	int p = 0;

	for(p = 0; p < 160; p++)
	{
		ii(p) = 21 + (p + 1);
		jj(p) = 181;
	}

	for(p = 160; p < 320; p++)
	{
		ii(p) = 181;
		jj(p) = 181 - ((p + 1) - 160);
	}

	for(p = 320; p < 480; p++)
	{
		ii(p) = 181 - ((p + 1) - 320);
		jj(p) = 21;
	}

	for(p = 480; p < num; p++)
	{
		ii(p) = 21;
		jj(p) = 21 + ((p + 1) - 480);
	}
}


/**********DEVICE FUNCTION DEFINITIONS***********/
__global__ void field_setup(kernel_ptr<float> const x, kernel_ptr<float> const y, kernel_ptr<float> fo)
{
	// Map from threadIdx / BlockIdx to pixel position

	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;

	if((i < NX) && (j < NY))
	{
		float value = 0.f;

		float rc = 0.015f;
		float rp = 0.005f;

		float sc = 0.03f;
		float sp = 0.05f;

		if (powf(x(i), 2) + powf(y(j), 2) <= powf(rc, 2))
		{
			value = sc;
		}

		if (powf(x(i) - rc * cos(-30 * (3.14159265f / 180)), 2) + powf(y(j) - rc * sin(30 * (3.14159265f / 180)), 2) <= powf(rp, 2))
		{
			value = sp;
		}

		if (powf(x(i) + rc * cos(-30 * (3.14159265f / 180)), 2) + powf(y(j) - rc * sin(30 * (3.14159265f / 180)), 2) <= powf(rp, 2))
		{
			value = sp;
		}

		if (powf(x(i), 2) + powf(y(j) + rc, 2) <= powf(rp, 2))
		{
			value = sp;
		}

		fo(i, j) = value;
	}
}

__global__ void propagation(
	kernel_ptr<int> const ii,
	kernel_ptr<int> const jj,
	kernel_ptr<float> const f,
	kernel_ptr<float> u,
	int k, int group_size, int Ng)
{
	// Map from threadIdx / BlockIdx to pixel position

	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;
	int g = threadIdx.z + blockIdx.z * blockDim.z;

	if(i < NX && j < NY && g < Ng) {
		float v = 1500.f * sqrtf(1.f + f(i, j));
		float r = v * DT / HX;
		float s = 2.f - 4.f * r * r;

		float val; // will hold new u at (i, j, k + 1)

		// not at boundary
		if (i != 0 && i != NX - 1 && j != 0 && j != NY - 1) {

			val =
				r * r *
				(u(i+1, j, k, g) +
				 u(i-1, j, k, g) +
				 u(i, j-1, k, g) +
				 u(i, j+1, k, g)) +
				s * u(i, j, k, g) -
				u(i, j, k-1, g);

			int p = g * group_size;

			int jp1 = jj(p);
			int jp2 = jj(p + group_size - 1);
			int ip1 = ii(p);
			int ip2 = ii(p + group_size - 1);

			minmax(jp1, jp2);
			minmax(ip1, ip2);

			// at sensor, k <= 24
			if (j + 1 >= jp1 && j + 1 <= jp2 && i + 1 >= ip1 && i + 1 <= ip2 && k + 1 <= 24) {
				float t = k * DT - TT;

				// add wave value
				val +=
					v * v * DT * DT *
					cosf(OMEGAC * t) *
					expf(-(t * t) / (2.f * TAO * TAO));
			}
		}

		// at boundary
		else {

			// boundary booleans
			bool top = (j == 0);
			bool bottom = (j == NY - 1);
			bool left = (i == 0);
			bool right = (i == NX - 1);

			// index variables for different boundary cases
			int ja = top ? (j + 1) : bottom ? (j - 1) : j;
			int jb = top ? (j + 2) : bottom ? (j - 2) : j;

			int ia = left ? (i + 1) : right ? (i - 1) : i;
			int ib = left ? (i + 2) : right ? (i - 2) : i;

			val =
				(2.f - 2.f * r - r * r) * u(i, j, k, g) +
				2.f * r * (1.f + r) * u(ia, ja, k, g) -
				r * r * u(ib, jb, k, g) +
				(2.f * r - 1.f) * u(i, j, k-1, g) -
				2.f * r * u(ia, ja, k-1, g);
		}

		u(i, j, k+1, g) = val;


	}
}

__global__ void propagation_at_corners(
	kernel_ptr<float> u,
	int Ng)
{
	int k = threadIdx.x + blockIdx.x * blockDim.x;
	int g = threadIdx.y + blockIdx.y * blockDim.y;

	if (k < NT && g < Ng) {
		u(0, 0, k, g) =
			1.f / 2.f * (u(0, 1, k, g) + u(1, 0, k, g));

		u(NX-1, 0, k, g) =
			1.f / 2.f * (u(NX-2, 0, k, g) + u(NX-1, 1, k, g));

		u(0, NY-1, k, g) =
			1.f / 2.f * (u(0, NY-2, k, g) + u(1, NY-1, k, g));

		u(NX-1, NY-1, k, g) =
			1.f / 2.f * (u(NX-2, NY-1, k, g) + u(NX-1, NY-2, k, g));
	}
}

__global__ void initial_signal(
	kernel_ptr<float> const u,
	kernel_ptr<float> g_bottom,
	kernel_ptr<float> g_right,
	kernel_ptr<float> g_top,
	kernel_ptr<float> g_left,
	int Ng)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int k = threadIdx.y + blockIdx.y * blockDim.y;
	int g = threadIdx.z + blockIdx.z * blockDim.z;

	if (i > 20 && i < 180 && k > 1 && k < NT && g < Ng) {
		// store values at bottom sensor row of u
		g_bottom(i, k, g) =
			u(i, 180, k, g);

		// store values at top sensor row of u
		g_top(i, k, g) =
			u(i, 20, k, g);



		// store values at right sensor column of u
		g_right(i, k, g) =
			u(180, i, k, g);


		// store values at left sensor column of u
		g_left(i, k, g) =
			u(20, i, k, g);
	}
}

__global__ void difference_signal(
	kernel_ptr<float> const u,
	kernel_ptr<float> const g_bottom,
	kernel_ptr<float> const g_right,
	kernel_ptr<float> const g_top,
	kernel_ptr<float> const g_left,
	kernel_ptr<float> rr_bottom,
	kernel_ptr<float> rr_right,
	kernel_ptr<float> rr_top,
	kernel_ptr<float> rr_left,
	int Ng)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int k = threadIdx.y + blockIdx.y * blockDim.y;
	int g = threadIdx.z + blockIdx.z * blockDim.z;

	if (i > 20 && i < 180 && k > 1 && k < NT && g < Ng) {
		// store difference at time k of original signal
		// and current signal at bottom sensor row
		rr_bottom(i, k, g) = g_bottom(i, k, g) - u(i, 180, k, g);

		// store difference at time k of original signal
		// and current signal at top sensor row
		rr_top(i, k, g) = g_top(i, k, g) - u(i, 20, k, g);

		// store difference at time k of original signal
		// and current signal at right sensor column
		rr_right(i, k, g) = g_right(i, k, g) - u(180, i, k, g);

		// store difference at time k of original signal
		// and current signal at left sensor column
		rr_left(i, k, g) = g_left(i, k, g) - u(20, i, k, g);
	}
}

__global__ void backpropagation1(
	kernel_ptr<float> z,
	kernel_ptr<float> const f,
	int k, int Ng)
{
	// Map from threadIdx / BlockIdx to pixel position

	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;
	int g = threadIdx.z + blockIdx.z * blockDim.z;

	if(i >= 1 && i < (NX - 1) && j >= 1 && j < (NY - 1) && g < Ng)
	{
		z(i, j, k, g) =
			1500.f * 1500.f * (DT * DT) *
			((1.f + f(i, j-1)) * z(i, j-1, k+1, g) +
			 (1.f + f(i, j+1)) * z(i, j+1, k+1, g) +
			 (1.f + f(i-1, j)) * z(i-1, j, k+1, g) +
			 (1.f + f(i+1, j)) * z(i+1, j, k+1, g) -
			 4.f * (1.f + f(i, j)) *
			 z(i, j, k+1, g)) / (H * H) +
			2.f * z(i, j, k+1, g) -
			z(i, j, k+2, g);
	}
}

__global__ void backpropagation2(
	kernel_ptr<float> z,
	kernel_ptr<float> const rr_bottom,
	kernel_ptr<float> const rr_right,
	kernel_ptr<float> const rr_top,
	kernel_ptr<float> const rr_left,
	int k, int Ng)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int g = threadIdx.y + blockIdx.y * blockDim.y;

	if (g < Ng) {
		if(i >= 21 && i < 180) {
			z(i, 180, k, g) =
				z(i, 179, k, g) +
				rr_bottom(i, k, g) * H * 1000.f;

			z(i, 20, k, g) =
				z(i, 21, k, g) +
				rr_top(i, k, g) * H * 1000.f;

			z(180, i, k, g) =
				z(179, i, k, g) +
				rr_right(i, k, g) * H * 1000.f;

			z(20, i, k, g) =
				z(21, i, k, g) +
				rr_left(i, k, g) * H * 1000.f;
		}

		if (i >= 1 && i < (NX - 1)) {
			z(i, 0, k, g) =
				z(i, 1, k, g);

			z(i, NY-1, k, g) =
				z(i, NY-2, k, g);

			z(0, i, k, g) =
				z(1, i, k, g);

			z(NX-1, i, k, g) =
				z(NX-2, i, k, g);
		}

		else if (i == 0) {
			z(0, 0, k, g) =
				(z(1, 0, k, g) +
				 z(0, 1, k, g)) / 2.f;

			z(NX-1, 0, k, g) =
				(z(NX-2, 0, k, g) +
				 z(NX-1, 1, k, g)) / 2.f;

			z(0, NY-1, k, g) =
				(z(1, NY-1, k, g) +
				 z(0, NY-2, k, g)) / 2.f;

			z(NX-1, NY-1, k, g) =
				(z(NX-2, NY-1, k, g) +
				 z(NX-1, NY-2, k, g)) / 2.f;
		}
	}
}

__global__ void laplace(
	kernel_ptr<float> const u,
	kernel_ptr<float> Lu,
	int Ng)
{
	// Map from threadIdx / BlockIdx to pixel position

	int tx = threadIdx.x + blockIdx.x * blockDim.x;
	int k = threadIdx.y + blockIdx.y * blockDim.y;
	int g = threadIdx.z + blockIdx.z * blockDim.z;

	if (tx < (NX * NY) && (k + 1) < NT && g < Ng) {
		int i = tx % NX;
		int j = tx / NX;

		int ja = (j > 0) ? (j - 1) : j;
		int jb = (j < NY - 1) ? (j + 1) : j;

		int ia = (i > 0) ? (i - 1) : i;
		int ib = (i < NX - 1) ? (i + 1) : i;

		Lu(i, j, k+1, g) =
			(u(i, ja, k+1, g) +
			 u(i, jb, k+1, g) +
			 u(ia, j, k+1, g) +
			 u(ib, j, k+1, g) -
			 4.f * u(i, j, k+1, g)) / (H * H);
	}
}

__global__ void laplace_corners(
	kernel_ptr<float> const u,
	kernel_ptr<float> Lu,
	int Ng)
{
	int k = threadIdx.x + blockIdx.x * blockDim.x;
	int g = threadIdx.y + blockIdx.y * blockDim.y;

	if ((k + 1) < NT && g < Ng) {
		Lu(0, 0, k+1, g) =
			(Lu(1, 0, k+1, g) +
			 Lu(0, 1, k+1, g)) / 2.f;

		Lu(NX-1, 0, k+1, g) =
			(Lu(NX-2, 0, k+1, g) +
			 Lu(NX-1, 1, k+1, g)) / 2.f;

		Lu(0, NY-1, k+1, g) =
			(Lu(1, NY-1, k+1, g) +
			 Lu(0, NY-2, k+1, g)) / 2.f;

		Lu(NX-1, NY-1, k+1, g) =
			(Lu(NX-2, NY-1, k+1, g) +
			 Lu(NX-1, NY-2, k+1, g)) / 2.f;
	}
}

__global__ void update_differential(
	kernel_ptr<float> df,
	kernel_ptr<float> const z,
	kernel_ptr<float> const Lu,
	kernel_ptr<float> const f,
	int Ng)
{
	// Map from threadIdx / BlockIdx to pixel position

	int tx = threadIdx.x + blockIdx.x * blockDim.x;
	int k = threadIdx.y + blockIdx.y * blockDim.y;
	int g = threadIdx.z + blockIdx.z * blockDim.z;

	if (tx < (NX * NY) && (k + 1) < NT && g < Ng) {
		int i = tx % NX;
		int j = tx / NX;

		atomicAdd(
			&df(i, j),
			z(i, j, k+1, g) *
			Lu(i, j, k+1, g) /
			(1.f + f(i, j)));
	}
}

__global__ void update_field(
	kernel_ptr<float> f,
	kernel_ptr<float> const df,
	kernel_ptr<float> f_minus_fo,
	kernel_ptr<float> const fo,
	float omega)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;

	if (i < NX && j < NY)
	{
		bool in_sensor_field = (i >= 21) && (i < 180) && (j >= 21) && (j < 180);

		float alpha = in_sensor_field ? 1.f : 0.f;

		f(i, j) += omega * alpha * df(i, j);
		f_minus_fo(i, j) = f(i, j) - fo(i, j);
	}
}

/**********INLINE FUNCTION DEFINITIONS**********/
inline int grid_size(int n, int threads)
{
	return ceil(float(n) / threads);
}


// POST-CONDITION: a <= b
template <typename T>
__host__ __device__ 
void minmax(T &a, T &b)
{
	if (a > b) {
		int t = a;
		a = b;
		b = t;
	}
}


