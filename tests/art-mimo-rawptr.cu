// HEADERS


#include <hip/hip_runtime.h>
#include <iostream>
#include <iomanip>
#include <string>
#include <limits>
#include <stdlib.h>
#include <fstream>
#include <math.h>
#include <time.h>

using namespace std;

// DEFINITIONS

#define NX 192				//was 201
#define PX 192				//was 224
#define NY 192				//was 201
#define PY 192				//was 224
#define NT 401

#define NS 640 							//number of sensors

#define BLOCK_X 16
#define BLOCK_Y 16

__constant__ float hx = 0.001f;
__constant__ float hy = 0.001f; // pixel size
__constant__ float h = 0.001f;

/* __constant__ float T = 1.3333e-04f; // 0.2f / 1500.f; */
__constant__ float dt = 3.3333e-07f; // T / 400.f;
/* __constant__ float fre = 125000.f; */
__constant__ float omegac = 7.8540e+05f; // 2.f * pi * fre; // wavelength
__constant__ float tao = 4.0000e-06f; // pi / omegac;
__constant__ float tt = 8.1573e-06f; // sqrtf(6.f * logf(2.f)) * tao; // time delay

// FUNCTIONS DECLARATION

void Ultrasonic_Tomography(int group_size, float target_epsilon, int max_iterations, int ti);
void Position_Transducers (int *&, int *&, int);
void IO_Files(float*, float*, float*, float*);
float norm(float*, int, int);

__global__ void field_setup(const float*, const float*, float*);
__global__ void propagation(int, int, int, int, const float*, float*, int);
__global__ void propagation_at_corners(float*);
__global__ void initial_signal(const float*, float*, float*, float*, float*, int);
__global__ void difference_signal(const float*, const float*, const float*, const float*, const float*, float*, float*, float*, float*, int);
__global__ void backpropagation1(float*, const float*, int);
__global__ void backpropagation2(float*, const float*, const float*, const float*, const float*, int);
__global__ void laplace(const float*, float*);
__global__ void laplace_corners(const float*, float*);
__global__ void update_differential(float*, const float*, const float*, const float*);
__global__ void update_field(float*, const float*, float*, const float*);

// MAIN PROGRAM

int main(int argc, char **argv)
{
	if (argc != 4) {
		cerr << "Usage: " << argv[0] << " <sensor group size> <target epsilon> <max iterations>\n\n";
		exit(1);
	}

	int group_size = stoi(argv[1]);
	float target_epsilon = stof(argv[2]);
	int max_iterations = stoi(argv[3]);

	if (max_iterations == -1)
		max_iterations = numeric_limits<int>::max();

	// Time measuring variables

	int ti = 0, tf = 0;

	// Function Execution

	// set floting-point precision on stdout and stderr
	cout << fixed << setprecision(10);
	cerr << fixed << setprecision(10);

	cout << "Ultrasonic Tomography Running:\n\n";

	ti = clock();
	cout << "ti = " << ti << "\n";

	Ultrasonic_Tomography(group_size, target_epsilon, max_iterations, ti);

	tf = clock();
	cout << "tf = " << tf << "\n"
		 << "tt = " << tf - ti << "\n"
		 << "Total Seconds = " << (float)(tf - ti) / CLOCKS_PER_SEC << "\n";

	// End of the program

	return 0;
}

inline
int grid_size(int n, int threads)
{
	return ceil(float(n) / threads);
}

// FUNCTIONS DEFINITION

void Ultrasonic_Tomography(int group_size, float target_epsilon, int max_iterations, int ti)
{
	// Simulation Variables

	float hx = 0.001f;
	float hy = 0.001f;

	int i = 0, j = 0, k = 0;

	float *x = new float[PX];
	float *y = new float[PY];
	float *fo = new float[PX * PY];

	// Kernel Preparation

	/*dim3 Grid_Size(13, 26);
	dim3 Block_Size(16, 8);*/

	/*dim3 Grid_Size(7, 51);
	dim3 Block_Size(32, 4);*/

	/*dim3 Grid_Size(7, 26);
	dim3 Block_Size(32, 8);*/

	dim3 Block_Size(BLOCK_X, BLOCK_Y);
	dim3 Grid_Size(grid_size(PX, BLOCK_X), grid_size(PY, BLOCK_Y));

	// Variables of allocation

	float *dev_x;
	int size_x = PX * sizeof(float);

	float *dev_y;
	int size_y = PX * sizeof(float);

	float *dev_fo;
	int size_fo = PX * PY * sizeof(float);

	float *dev_u;
	int size_u = PX * PY * NT * sizeof(float);


	float *dev_g1;
	float *dev_g2;
	float *dev_g3;
	float *dev_g4;
	int size_g = PX * NT * (NS / group_size) * sizeof(float);

	hipMalloc((void**) &dev_x, size_x);
	hipMalloc((void**) &dev_y, size_y);
	hipMalloc((void**) &dev_fo, size_fo);
	hipMalloc((void**) &dev_u, size_u);
	hipMalloc((void**) &dev_g1, size_g);
	hipMalloc((void**) &dev_g2, size_g);
	hipMalloc((void**) &dev_g3, size_g);
	hipMalloc((void**) &dev_g4, size_g);

	hipMemset(dev_u, 0.f, size_u);
	hipMemset(dev_g1, 0.f, size_g);
	hipMemset(dev_g2, 0.f, size_g);
	hipMemset(dev_g3, 0.f, size_g);
	hipMemset(dev_g4, 0.f, size_g);

	// Environment Initialization

	for(i = 0; i < NX; i++)
	{
		x[i] = -0.1f + i * hx;
	}

	for(j = 0; j < NY; j++)
	{
		y[j] = -0.1f + j * hy;
	}

	hipMemcpy(dev_x, x, size_x, hipMemcpyHostToDevice);
	hipMemcpy(dev_y, y, size_y, hipMemcpyHostToDevice);

	field_setup<<<Grid_Size, Block_Size>>>(dev_x, dev_y, dev_fo);

	hipMemcpy(fo, dev_fo, size_fo, hipMemcpyDeviceToHost);

	// Position of the transducers

	int *ii, *jj;
	Position_Transducers (ii, jj, NS);

	dim3 threads_propagation(16, 16);
	dim3 grid_propagation(grid_size(PX, threads_propagation.x),
							grid_size(PY, threads_propagation.y));

	int p;
	for(p = 0; p < NS; p += group_size)
	{
		hipMemset(dev_u, 0.f, size_u);

		int jp1 = jj[p];
		int jp2 = jj[p + group_size - 1];
		int ip1 = ii[p];
		int ip2 = ii[p + group_size - 1];

		if (jp2 < jp1)
		{
			int jp = jp1;
			jp1 = jp2;
			jp2 = jp;
		}

		if (ip2 < ip1)
		{
			int ip = ip1;
			ip1 = ip2;
			ip2 = ip;
		}

		// Boundary

		for(k = 1; k < NT - 1; k++)
		{
			propagation<<<grid_propagation, threads_propagation>>>(jp1, jp2, ip1, ip2, dev_fo, dev_u, k);
		}

		// Four corners

		propagation_at_corners<<<1, NT>>>(dev_u);

		initial_signal<<<NT - 2, 159>>>(dev_u, dev_g1, dev_g2, dev_g3, dev_g4, p / group_size);
	}

	// Kaczmarz method
	// propagation

	float *dev_rr1;
	int size_rr1 = PX * NT * sizeof(float);

	float *dev_rr2;
	int size_rr2 = PX * NT * sizeof(float);

	float *dev_rr3;
	int size_rr3 = PX * NT * sizeof(float);

	float *dev_rr4;
	int size_rr4 = PX * NT * sizeof(float);

	float *dev_z;
	int size_z = PX * PY * (NT + 1) * sizeof(float);

	float *dev_Lu;
	int size_Lu = PX * PY * NT * sizeof(float);

	float *dev_f;
	int size_f = PX * PY * sizeof(float);

	float *dev_df;
	int size_df = PX * PY * sizeof(float);

	float *dev_f_minus_fo;
	int size_f_minus_fo = PX * PY * sizeof(float);

	// Allocation

	hipMalloc((void**) &dev_rr1, size_rr1);
	hipMalloc((void**) &dev_rr2, size_rr2);
	hipMalloc((void**) &dev_rr3, size_rr3);
	hipMalloc((void**) &dev_rr4, size_rr4);
	hipMalloc((void**) &dev_z, size_z);
	hipMalloc((void**) &dev_Lu, size_Lu);
	hipMalloc((void**) &dev_f, size_f);
	hipMalloc((void**) &dev_df, size_df);
	hipMalloc((void**) &dev_f_minus_fo, size_f_minus_fo);

	hipMemset(dev_rr1, 0.f, size_rr1);
	hipMemset(dev_rr2, 0.f, size_rr2);
	hipMemset(dev_rr3, 0.f, size_rr3);
	hipMemset(dev_rr4, 0.f, size_rr4);
	hipMemset(dev_f, 0.f, size_f);
	hipMemset(dev_Lu, 0.f, size_Lu);

	float *f = new float[PX * PY];
	float *f_minus_fo = new float[PX * PY];

	// initialize epsilon values
	float prev_epsilon = std::numeric_limits<float>::infinity();
	float curr_epsilon = -std::numeric_limits<float>::infinity();

	cerr << "writing convergence to 'art_convergence.txt'...\n"
		 << "writing time to 'art_time.txt'...\n";

	ofstream convergence_file("art_convergence.txt");
	ofstream time_file("art_time.txt");

	dim3 threads_backpropagation1(16, 16);
	dim3 grid_backpropagation1(grid_size(PX, threads_backpropagation1.x),
								grid_size(PY, threads_backpropagation1.y));

	dim3 threads_laplace(16, 16, 1);
	dim3 grid_laplace(grid_size(PX, threads_laplace.x),
						grid_size(PY, threads_laplace.y),
						grid_size(NT, threads_laplace.z));

	dim3 threads_differential(16, 16, 1);
	dim3 grid_differential(grid_size(PX, threads_differential.x),
							grid_size(PY, threads_differential.y),
							grid_size(NT, threads_differential.z));

	for(int iter = 0; iter < max_iterations; iter++)
	{
		cout << "\nIter: " << iter << "\n";
		hipMemset(dev_u, 0.f, size_u);

		for(p = 0; p < NS; p += group_size)
		{
			int jp1 = jj[p];
			int jp2 = jj[p + group_size - 1];
			int ip1 = ii[p];
			int ip2 = ii[p + group_size - 1];

			if (jp2 < jp1)
			{
				int jp = jp1;
				jp1 = jp2;
				jp2 = jp;
			}

			if (ip2 < ip1)
			{
				int ip = ip1;
				ip1 = ip2;
				ip2 = ip;
			}

			// Boundary

			for(k = 1; k < NT - 1; k++)
			{
				propagation<<<grid_propagation, threads_propagation>>>(jp1, jp2, ip1, ip2, dev_f, dev_u, k);
			}

			// Four corners

			propagation_at_corners<<<1, NT>>>(dev_u);
			difference_signal<<<NT - 2, 159>>>(dev_u, dev_g1, dev_g2, dev_g3, dev_g4, dev_rr1, dev_rr2, dev_rr3, dev_rr4, p / group_size);

			hipMemset(dev_z, 0.f, size_z);

			for(k = NT - 2; k > 0; k--)
			{
				backpropagation1<<<grid_backpropagation1, threads_backpropagation1>>>(dev_z, dev_f, k);
				backpropagation2<<<1, NX>>>(dev_z, dev_rr1, dev_rr2, dev_rr3, dev_rr4, k);
			}

			laplace<<<grid_laplace, threads_laplace>>>(dev_u, dev_Lu);
			laplace_corners<<<grid_size(NT, 32), 32>>>(dev_u, dev_Lu);

			hipMemset(dev_df, 0.f, size_df);
			update_differential<<<grid_differential, threads_differential>>>(dev_df, dev_z, dev_Lu, dev_f);

			update_field<<<Grid_Size, Block_Size>>>(dev_f, dev_df, dev_f_minus_fo, dev_fo);
		}

		hipMemcpy(f_minus_fo, dev_f_minus_fo, size_f_minus_fo, hipMemcpyDeviceToHost);

		curr_epsilon = norm(f_minus_fo, NX, NY) / norm(fo, NX, NY) * 100.f;
		float current_t = (float)(clock()-ti) / CLOCKS_PER_SEC;

		convergence_file << curr_epsilon << " ";
		time_file << current_t << " ";

		cout << "epsilon = " << curr_epsilon << "\n";

		// stop if reached target epsilon
		if (curr_epsilon <= target_epsilon) {
			break;
		}

		// stop if epsilon diverges
		if (curr_epsilon > prev_epsilon ||
				std::isnan(curr_epsilon)) {
			break;
		}

		// update prev_epsilon
		prev_epsilon = curr_epsilon;
	}

	hipMemcpy(f, dev_f, size_f, hipMemcpyDeviceToHost);

	IO_Files(x, y, fo, f);

	// Free Variables

	hipFree(dev_x);
	hipFree(dev_y);
	hipFree(dev_fo);
	hipFree(dev_u);
	hipFree(dev_g1);
	hipFree(dev_g2);
	hipFree(dev_g3);
	hipFree(dev_g4);
	hipFree(dev_rr1);
	hipFree(dev_rr2);
	hipFree(dev_rr3);
	hipFree(dev_rr4);
	hipFree(dev_z);
	hipFree(dev_Lu);
	hipFree(dev_f);
	hipFree(dev_df);
	hipFree(dev_f_minus_fo);

	delete [] x;
	delete [] y;
	delete [] fo;
	delete [] ii;
	delete [] jj;
	delete [] f;
	delete [] f_minus_fo;

	hipDeviceReset();
}
void Position_Transducers(int *&ii, int *&jj, int num)
{
//returns the (x,y) coordinates of the number of total transducers
	int p = 0;
	ii = (int*)malloc(num * sizeof(int));
	jj = (int*)malloc(num * sizeof(int));


	for(p = 0; p < NS/4; p++)
	{
		ii[p] = 21 + (p + 1);
		jj[p] = 181;
	}

	for(p = 160; p < 320; p++)
	{
		ii[p] = 181;
		jj[p] = 181 - ((p + 1) - 160);
	}

	for(p = 320; p < 480; p++)
	{
		ii[p] = 181 - ((p + 1) - 320);
		jj[p] = 21;
	}

	for(p = 480; p < num; p++)
	{
		ii[p] = 21;
		jj[p] = 21 + ((p + 1) - 480);
	}
}
__global__ void field_setup(const float *x, const float *y, float *fo)
{
	// Map from threadIdx / BlockIdx to pixel position

	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;

	if((i < NX) && (j < NY))
	{
		int offset = i + PX * j;
		float value = 0.f;

		/* if(((sqrtf(powf(x[i] - 0.015f, 2.f) + powf(y[j] + 0.000f, 2.f))) <= 0.005f) || ((sqrtf(powf(x[i] + 0.015f, 2.f) + powf(y[j] + 0.000f, 2.f))) <= 0.005f)) */
		/* { */
		/*	value = 0.06f; */
		/* } */
		/* else */
		/* { */
		/*	if(sqrtf(x[i] * x[i] + y[j] * y[j]) <= 0.03f) */
		/*	{ */
		/*		value = 0.02f; */
		/*	} */
		/*	else */
		/*	{ */
		/*		value = 0.f; */
		/*	} */
		/* } */

		float rc = 0.015f;
		float rp = 0.005f;
		/* float lim = 0.020f; */

		float sc = 0.03f;
		float sp = 0.05f;
		/* float sb = 0.02f; */

		if (powf(x[i], 2) + powf(y[j], 2) <= powf(rc, 2))
		{
			value = sc;
		}

		if (powf(x[i] - rc * cos(-30 * (3.14159265f / 180)), 2) + powf(y[j] - rc * sin(30 * (3.14159265f / 180)), 2) <= powf(rp, 2))
		{
			value = sp;
		}

		if (powf(x[i] + rc * cos(-30 * (3.14159265f / 180)), 2) + powf(y[j] - rc * sin(30 * (3.14159265f / 180)), 2) <= powf(rp, 2))
		{
			value = sp;
		}

		if (powf(x[i], 2) + powf(y[j] + rc, 2) <= powf(rp, 2))
		{
			value = sp;
		}

		fo[offset] = value;


		/*int offset = i + NX * j;
		float value = 0.f;

		if (((sqrtf(powf(x[i] - 0.05f, 2.f) + powf(y[j] + 0.000f, 2.f))) <= 0.005f) || ((sqrtf(powf(x[i] + 0.05f, 2.f) + powf(y[j] + 0.000f, 2.f))) <= 0.005f))
		{
			value = 0.06f;
		}
		else
		{
			if (sqrtf(x[i] * x[i] + y[j] * y[j]) <= 0.03f)
			{
				value = 0.02f;
			}
			else
			{
				if ((x[i] >= -0.05f) && (x[i] <= 0.05f) && (y[j] >= -0.06f) && (y[j] <= -0.045f))
				{
					value = 0.04f;
				}
				else
				{
					if ((x[i] >= -0.03f) && (x[i] <= 0.00f) && (y[j] <= 0.065f) && (y[j] >= (0.04f - 0.5f * x[i])))
					{
						value = 0.03f;
					}
					else
					{
						if ((x[i] >= 0.00f) && (x[i] <= 0.03f) && (y[j] <= 0.065f) && (y[j] >= (0.04f + 0.5f * x[i])))
						{
							value = 0.03f;
						}
						else
						{
							value = 0.f;
						}
					}
				}
			}
		}

		fo[offset] = value;
		v[offset] = 1500.f * sqrtf(1.f + value);
		r[offset] = v[offset] * dt / hx;
		r2[offset] = powf(r[offset], 2.f);
		s[offset] = 2.f - 4.f * r2[offset];
		*/
	}
}

__global__ void propagation(int jp1, int jp2, int ip1, int ip2, const float *f, float *u, int k)
{
	// Map from threadIdx / BlockIdx to pixel position

	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;

	if(i < NX && j < NY) {
		float v = 1500.f * sqrtf(1.f + f[i + PX * j]);
		float r = v * dt / hx;
		float s = 2.f - 4.f * r * r;

		float val; // wil hold new u at (i, j, k + 1)

		// not at boundary
		if (i != 0 && i != NX - 1 && j != 0 && j != NY - 1) {

			// update val
			val =
				r * r *
				(u[(i + 1) + PX * j + PX * PY * k] +
				 u[(i - 1) + PX * j + PX * PY * k] +
				 u[i + PX * (j - 1) + PX * PY * k] +
				 u[i + PX * (j + 1) + PX * PY * k]) +
				s * u[i + PX * j + PX * PY * k] -
				u[i + PX * j + PX * PY * (k - 1)];

			// at sensor, k <= 24
			if ((j + 1) >= jp1 && (j + 1) <= jp2 && (i + 1) >= ip1 && (i + 1) <= ip2 && (k + 1) <= 24) {
				float t = k * dt - tt;

				// add wave value
				val +=
					v * v * dt * dt *
					cosf(omegac * t) *
					expf(-(t * t) / (2.f * tao * tao));
			}
		}

		// at boundary
		else {

			// index variables for different boundary cases
			// TODO: need better names
			int i_A, i_B, j_A, j_B;

			// left boundary
			if (i == 0)
			{
				i_A = i + 1;
				j_A = j;
				i_B = i + 2;
				j_B = j;
			}

			// right boundary
			else if (i == NX - 1)
			{
				i_A = i - 1;
				j_A = j;
				i_B = i - 2;
				j_B = j;
			}

			// top boundary
			else if (j == 0)
			{
				i_A = i;
				j_A = j + 1;
				i_B = i;
				j_B = j + 2;
			}

			// bottom boundary
			else
			{
				i_A = i;
				j_A = j - 1;
				i_B = i;
				j_B = j - 2;
			}

			val =
				(2.f - 2.f * r - r * r) * u[i + PX * j + PX * PY * k] +
				2.f * r * (1.f + r) * u[i_A + PX * j_A + PX * PY * k] -
				r * r * u[i_B + PX * j_B + PX * PY * k] +
				(2.f * r - 1.f) * u[i + PX * j + PX * PY * (k - 1)] -
				2.f * r * u[i_A + PX * j_A + PX * PY * (k - 1)];
		}

		// update u at (i, j, k + 1)
		u[i + PX * j + PX * PY * (k + 1)] = val;
	}
}

__global__ void propagation_at_corners(float *u)
{
	int k = threadIdx.x;

	u[PX * PY * k] =
		1.f / 2.f * (u[PX + k] + u[1 + k]);

	u[(NX - 1) + PX * PY * k] =
		1.f / 2.f * (u[(NX - 2) + PX * PY * k] + u[(NX - 1) + PX + PX * PY * k]);

	u[PX * (NY - 1) + PX * PY * k] =
		1.f / 2.f * (u[PX * (NY - 2) + PX * PY * k] + u[1 + PX * (NY - 1) + PX * PY * k]);

	u[(NX - 1) + PX * (NY - 1) + PX * PY * k] =
		1.f / 2.f * (u[(NX - 2) + PX * (NY - 1) + PX * PY * k] + u[(NX - 1) + PX * (NY - 2) + PX * PY * k]);
}

__global__ void initial_signal(const float *u, float *g1, float *g2, float *g3, float *g4, int p)
{
	int i = threadIdx.x;
	int k = blockIdx.x;

	g1[(i + 21) + PX * (k + 2) + PX * NT * p] =
		u[(i + 21) + PX * 180 + PX * PY * (k + 2)];

	g3[(i + 21) + PX * (k + 2) + PX * NT * p] =
		u[(i + 21) + PX * 20 + PX * PY * (k + 2)];

	g2[(i + 21) + PX * (k + 2) + PX * NT * p] =
		u[180 + PX * (i + 21) + PX * PY * (k + 2)];

	g4[(i + 21) + PX * (k + 2) + PX * NT * p] =
		u[20 + PX * (i + 21) + PX * PY * (k + 2)];
}

__global__ void difference_signal(const float *u, const float *g1, const float *g2, const float *g3, const float *g4, float *rr1, float *rr2, float *rr3, float *rr4, int p)
{
	int i = threadIdx.x;
	int k = blockIdx.x;

	rr1[(i + 21) + PX * (k + 2)] =
		g1[(i + 21) + PX * (k + 2) + PX * NT * p] -
		u[(i + 21) + PX * 180 + PX * PY * (k + 2)];

	rr3[(i + 21) + PX * (k + 2)] =
		g3[(i + 21) + PX * (k + 2) + PX * NT * p] -
		u[(i + 21) + PX * 20 + PX * PY * (k + 2)];

	rr2[(i + 21) + PX * (k + 2)] =
		g2[(i + 21) + PX * (k + 2) + PX * NT * p] -
		u[180 + PX * (i + 21) + PX * PY * (k + 2)];

	rr4[(i + 21) + PX * (k + 2)] =
		g4[(i + 21) + PX * (k + 2) + PX * NT * p] -
		u[20 + PX * (i + 21) + PX * PY * (k + 2)];
}

__global__ void backpropagation1(float *z, const float *f, int k)
{
	// Map from threadIdx / BlockIdx to pixel position

	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;

	if((i >= 1) && (i < (NX - 1)) && (j >= 1) && (j < (NY - 1)))
	{
		z[i + PX * j + PX * PY * k] =
			1500.f * 1500.f * (dt * dt) *
			((1.f + f[i + PX * (j - 1)]) * z[i + PX * (j - 1) + PX * PY * (k + 1)] +
			 (1.f + f[i + PX * (j + 1)]) * z[i + PX * (j + 1) + PX * PY * (k + 1)] +
			 (1.f + f[(i - 1) + PX * j]) * z[(i - 1) + PX * j + PX * PY * (k + 1)] +
			 (1.f + f[(i + 1) + PX * j]) * z[(i + 1) + PX * j + PX * PY * (k + 1)] -
			 4.f * (1.f + f[i + PX * j]) *
			 z[i + PX * j + PX * PY * (k + 1)]) / (h * h) +
			2.f * z[i + PX * j + PX * PY * (k + 1)] -
			z[i + PX * j + PX * PY * (k + 2)];
	}
}

__global__ void backpropagation2(float *z, const float *rr1, const float *rr2, const float *rr3, const float *rr4, int k)
{
	int i = threadIdx.x;

	if((i >= 21) && (i < 180))
	{
		z[i + PX * 180 + PX * PY * k] =
		// z[k][180][i]
			z[i + PX * 179 + PX * PY * k] +
			// z[k][179][i]
			rr1[i + PX * k] * h * 1000.f;
			// rr1[k][i]

		z[i + PX * 20 + PX * PY * k] =
		// z[k][20][i]
			z[i + PX * 21 + PX * PY * k] +
			// z[k][21][i]
			rr3[i + PX * k] * h * 1000.f;
			// z[k][i]

		z[180 + PX * i + PX * PY * k] =
		// z[k][i][180]
			z[179 + PX * i + PX * PY * k] +
			// z[k][i][179]
			rr2[i + PX * k] * h * 1000.f;
			// rr2[k][i]

		z[20 + PX * i + PX * PY * k] =
		// z[k][i][20]
			z[21 + PX * i + PX * PY * k] +
			// z[k][i][21]
			rr4[i + PX * k] * h * 1000.f;
			// rr4[k][i]
	}

	if((i >= 1) && (i < (NX - 1)))
	{
		z[i + PX * PY * k] =
		// z[k][0][i]
			z[i + PX + PX * PY * k];
			// z[k][1][i]

		z[i + PX * (NY - 1) + PX * PY * k] =
		// z[k][NY - 1][i]
			z[i + PX * (NY - 2) + PX * PY * k];
			// z[k][NY - 2][i]

		z[PX * i + PX * PY * k] =
		// z[k][i][0]
			z[1 + PX * i + PX * PY * k];
			// z[k][i][1]

		z[(NX - 1) + PX * i + PX * PY * k] =
		// z[k][i][NX - 1]
			z[(NX - 2) + PX * i + PX * PY * k];
			// z[k][i][NX - 2]
	}

	else if(i == 0)
	{
		z[PX * PY * k] =
			(z[1 + PX * PY * k] +
			 z[PX + PX * PY * k]) / 2.f;
			// z[k][1][0]

		z[(NX - 1) + PX * PY * k] =
			(z[(NX - 2) + PX * PY * k] +
			 z[(NX - 1) + PX + PX * PY * k]) / 2.f;
			// z[k][1][NX - 1]

		z[PX * (NY - 1) + PX * PY * k] =
			(z[1 + PX * (NY - 1) + PX * PY * k] +
			// z[k][NY - 1][1]
			 z[PX * (NY - 2) + PX * PY * k]) / 2.f;
			// z[k][NY - 2][0]

		z[(NX - 1) + PX * (NY - 1) + PX * PY * k] =
			(z[(NX - 2) + PX * (NY - 1) + PX * PY * k] +
			// z[k][NY - 1][NX - 2]
			 z[(NX - 1) + PX * (NY - 2) + PX * PY * k]) / 2.f;
			// z[k][NY - 2][NX - 1]
	}
}

__global__ void laplace(const float *u, float *Lu)
{
	// Map from threadIdx / BlockIdx to pixel position

	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;
	int k = threadIdx.z + blockIdx.z * blockDim.z;

	if (i < NX && j < NY && (k + 1) < NT) {

		int j_prev = (j > 0) ? j - 1 : j;
		int j_next = (j < NY - 1) ? j + 1 : j;

		int i_prev = (i > 0) ? i - 1 : i;
		int i_next = (i < NX - 1) ? i + 1 : i;

		Lu[i + PX * j + PX * PY * (k + 1)] =
			(u[i + PX * j_prev + PX * PY * (k + 1)] +
			 u[i + PX * j_next + PX * PY * (k + 1)] +
			 u[i_prev + PX * j + PX * PY * (k + 1)] +
			 u[i_next + PX * j + PX * PY * (k + 1)] -
			 4.f * u[i + PX * j + PX * PY * (k + 1)]) / (h * h);
	}
}

__global__ void laplace_corners(const float *u, float *Lu)
{
	int k = threadIdx.x + blockIdx.x * blockDim.x;

	if ((k + 1) < NT) {
		Lu[PX * PY * (k + 1)] =
			(Lu[1 + PX * PY * (k + 1)] +
			 Lu[PX + PX * PY * (k + 1)]) / 2.f;

		Lu[(NX - 1) + PX * PY * (k + 1)] =
			(Lu[(NX - 2) + PX * PY * (k + 1)] +
			 Lu[(NX - 1) + PX + PX * PY * (k + 1)]) / 2.f;

		Lu[PX * (NY - 1) + PX * PY * (k + 1)] =
			(Lu[1 + PX * (NY - 1) + PX * PY * (k + 1)] +
			 Lu[PX * (NY - 2) + PX * PY * (k + 1)]) / 2.f;

		Lu[(NX - 1) + PX * (NY - 1) + PX * PY * (k + 1)] =
			(Lu[(NX - 2) + PX * (NY - 1) + PX * PY * (k + 1)] +
			 Lu[(NX - 1) + PX * (NY - 2) + PX * PY * (k + 1)]) / 2.f;
	}
}

__global__ void update_differential(float *df, const float *z, const float *Lu, const float *f)
{
	// Map from threadIdx / BlockIdx to pixel position

	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;
	int k = threadIdx.z + blockIdx.z * blockDim.z;

	if(i < NX && j < NY && (k + 1) < NT) {

		atomicAdd(&df[i + PX * j],
					z[i + PX * j + PX * PY * (k + 1)] *
					Lu[i + PX * j + PX * PY * (k + 1)] /
					(1.f + f[i + PX * j]));
	}
}

__global__ void update_field(float *f, const float *df, float *f_minus_fo, const float *fo)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;

	if((i < NX) && (j < NY))
	{
		int offset = i + PX * j;

		bool in_sensor_field = (i >= 21) && (i < 180) && (j >= 21) && (j < 180);

		float alpha = in_sensor_field ? 1.f : 0.f;

		f[offset] += 20000.f * alpha * df[offset];
		f_minus_fo[offset] = f[offset] - fo[offset];
	}
}

void IO_Files(float *x, float *y, float *fo, float *f)
{
	int i = 0, j = 0;

	// I/O Files

	ofstream x_file, y_file;
	ofstream fo_file;
	ofstream f_file;

	cerr << "writing x to 'dev_x.txt'...\n"
		 << "writing y to 'dev_y.txt'...\n"
		 << "writing f0 to 'dev_f0.txt'...\n"
		 << "writing f to 'dev_f.txt'...\n\n";

	x_file.open("dev_x.txt");
	y_file.open("dev_y.txt");
	fo_file.open("dev_f0.txt");
	f_file.open("dev_f.txt");

	for(i = 0; i < NX; i++)
	{
		x_file << x[i];
		x_file << "\n";
	}

	for(j = 0; j < NX; j++)
	{
		y_file << y[j];
		y_file << "\n";
	}

	for(j = 0; j < NY; j++)
	{
		for(i = 0; i < NX; i++)
		{
			fo_file << fo[i + PX * j];
			fo_file << " ";
		}

		fo_file << "\n";
	}

	for(j = 0; j < NY; j++)
	{
		for(i = 0; i < NX; i++)
		{
			f_file << f[i + PX * j];
			f_file << " ";
		}

		f_file << "\n";
	}

	x_file.close();
	y_file.close();
	fo_file.close();
	f_file.close();
}

float norm(float *A, int nx, int ny)
{
	float sum = 0;

	for (int j = 0; j < ny; ++j)
		for (int i = 0; i < nx; ++i)
			sum += A[i + PX * j] * A[i + PX * j];

	return sqrtf(sum);
}
