// HEADERS


#include <hip/hip_runtime.h>
#include <iostream>
#include <stdlib.h>
#include <fstream>
#include <math.h>
#include <time.h>

using namespace std;

// DEFINITIONS

#define NX 201
#define PX 224
#define NY 201
#define PY 224
#define NT 401
/* #define NT 416 */

#define BLOCK_X 16
#define BLOCK_Y 16

#define SENSOR_GROUP_SIZE 10

#ifndef I_MAX
#define I_MAX 20
#endif /* I_MAX */

__constant__ float hx = 0.001f;
__constant__ float hy = 0.001f; // pixel size
__constant__ float h = 0.001f;

/* __constant__ float T = 1.3333e-04f; // 0.2f / 1500.f; */
__constant__ float dt = 3.3333e-07f; // T / 400.f;
/* __constant__ float fre = 125000.f; */
__constant__ float omegac = 7.8540e+05f; // 2.f * pi * fre; // wavelength
__constant__ float tao = 4.0000e-06f; // pi / omegac;
__constant__ float tt = 8.1573e-06f; // sqrtf(6.f * logf(2.f)) * tao; // time delay

// FUNCTIONS DECLARATION

void Ultrasonic_Tomography();
void IO_Files(float*, float*, float*, float*);
float norm(float*, int, int);

__global__ void field_setup(const float*, const float*, float*);
__global__ void propagation(int, int, int, int, const float*, float*, int);
__global__ void propagation_at_corners(float*);
__global__ void initial_signal(const float*, float*, float*, float*, float*, int);
__global__ void difference_signal(const float*, const float*, const float*, const float*, const float*, float*, float*, float*, float*, int);
__global__ void backpropagation1(float*, const float*, int);
__global__ void backpropagation2(float*, const float*, const float*, const float*, const float*, int);
__global__ void laplace1(const float*, float*);
__global__ void laplace2(const float*, float*);
__global__ void update_differential(float*, const float*, const float*, const float*);
__global__ void update_field(float*, const float*, float*, const float*);

// MAIN PROGRAM

int main(void)
{
	// Time measuring variables

	int ti = 0, tf = 0;

	// Function Execution

	printf("Ultrasonic Tomography Running:\n\n");

	ti = clock();
	printf("ti = %d\n", ti);

	Ultrasonic_Tomography();

	tf = clock();
	printf("tf = %d\n", tf);
	printf("tt = %d\n", tf - ti);
	printf("Total Seconds = %f\n", (float)(tf - ti)  / CLOCKS_PER_SEC);

	hipDeviceReset();

	// End of the program

	/* system("pause"); */
	return 0;
}

inline 
int grid_size(int n, int threads)
{
    return ceil(float(n) / threads);
}

// FUNCTIONS DEFINITION

void Ultrasonic_Tomography()
{
	// Simulation Variables

	float hx = 0.001f;
	float hy = 0.001f;

	int i = 0, j = 0, k = 0;
	int Nx_Ny = PX * PY;
	int Nx_Ny_Nt = PX * PY * NT;
	int Nx_Nt = PX * NT;

	float *x = new float[PX];
	float *y = new float[PY];
	float *fo = new float[Nx_Ny];
	float *u = new float[Nx_Ny_Nt];

	// Kernel Preparation

	/*dim3 Grid_Size(13, 26);
	dim3 Block_Size(16, 8);*/

	/*dim3 Grid_Size(7, 51);
	dim3 Block_Size(32, 4);*/

	/*dim3 Grid_Size(7, 26);
	dim3 Block_Size(32, 8);*/

	dim3 Block_Size(BLOCK_X, BLOCK_Y);
	dim3 Grid_Size(grid_size(PX, BLOCK_X), grid_size(PY, BLOCK_Y));

	// Variables of allocation

	float *dev_x;
	int size_x = PX * sizeof(float);

	float *dev_y;
	int size_y = PX * sizeof(float);

	float *dev_fo;
	int size_fo = Nx_Ny * sizeof(float);

	float *dev_u;
	int size_u = Nx_Ny_Nt * sizeof(float);

	float *dev_g1;
	int size_g1 = Nx_Nt * 640 * sizeof(float);

	float *dev_g2;
	int size_g2 = Nx_Nt * 640 * sizeof(float);

	float *dev_g3;
	int size_g3 = Nx_Nt * 640 * sizeof(float);

	float *dev_g4;
	int size_g4 = Nx_Nt * 640 * sizeof(float);

	hipMalloc((void**) &dev_x, size_x);
	hipMalloc((void**) &dev_y, size_y);
	hipMalloc((void**) &dev_fo, size_fo);
	hipMalloc((void**) &dev_u, size_u);
	hipMalloc((void**) &dev_g1, size_g1);
	hipMalloc((void**) &dev_g2, size_g2);
	hipMalloc((void**) &dev_g3, size_g3);
	hipMalloc((void**) &dev_g4, size_g4);

	hipMemset(dev_u, 0.f, size_u);
	hipMemset(dev_g1, 0.f, size_g1);
	hipMemset(dev_g2, 0.f, size_g2);
	hipMemset(dev_g3, 0.f, size_g3);
	hipMemset(dev_g4, 0.f, size_g4);

	// Environment Initialization

	for(i = 0; i < NX; i++)
	{
		x[i] = -0.1f + i * hx;
	}

	for(j = 0; j < NY; j++)
	{
		y[j] = -0.1f + j * hy;
	}

	hipMemcpy(dev_x, x, size_x, hipMemcpyHostToDevice);
	hipMemcpy(dev_y, y, size_y, hipMemcpyHostToDevice);

	field_setup<<<Grid_Size, Block_Size>>>(dev_x, dev_y, dev_fo);

	hipMemcpy(fo, dev_fo, size_fo, hipMemcpyDeviceToHost);

	// Position of the transducers

	int p = 0;
	int *jj = new int[640];
	int *ii = new int[640];

	for(p = 0; p < 160; p++)
	{
		ii[p] = 21 + (p + 1);
		jj[p] = 181;
	}

	for(p = 160; p < 320; p++)
	{
		ii[p] = 181;
		jj[p] = 181 - ((p + 1) - 160);
	}

	for(p = 320; p < 480; p++)
	{
		ii[p] = 181 - ((p + 1) - 320);
		jj[p] = 21;
	}

	for(p = 480; p < 640; p++)
	{
		ii[p] = 21;
		jj[p] = 21 + ((p + 1) - 480);
	}

	for(p = 0; p < 640; p += SENSOR_GROUP_SIZE)
	{
		hipMemset(dev_u, 0.f, size_u);

		int jp1 = jj[p];
		int jp2 = jj[p + SENSOR_GROUP_SIZE - 1];
		int ip1 = ii[p];
		int ip2 = ii[p + SENSOR_GROUP_SIZE - 1];

		if (jp2 < jp1)
		{
			int jp = jp1;
			jp1 = jp2;
			jp2 = jp;
		}

		if (ip2 < ip1)
		{
			int ip = ip1;
			ip1 = ip2;
			ip2 = ip;
		}

		// Boundary

		for(k = 1; k < NT - 1; k++)
		{
			propagation<<<Grid_Size, Block_Size>>>(jp1, jp2, ip1, ip2, dev_fo, dev_u, k);
		}

		// Four corners

		propagation_at_corners<<<1, NT>>>(dev_u);

		initial_signal<<<NT - 2, 159>>>(dev_u, dev_g1, dev_g2, dev_g3, dev_g4, p);
	}

	// Kaczmarz method
	// propagation

	/* float *f_t = new float[Nx_Ny * I_MAX]; */

	float *dev_rr1;
	int size_rr1 = Nx_Nt * sizeof(float);

	float *dev_rr2;
	int size_rr2 = Nx_Nt * sizeof(float);

	float *dev_rr3;
	int size_rr3 = Nx_Nt * sizeof(float);

	float *dev_rr4;
	int size_rr4 = Nx_Nt * sizeof(float);

	float *dev_z;
	int size_z = Nx_Ny * (NT + 1) * sizeof(float);

	float *dev_Lu;
	int size_Lu = Nx_Ny_Nt * sizeof(float);

	float *dev_f;
	int size_f = Nx_Ny * sizeof(float);

	float *dev_df;
	int size_df = Nx_Ny * sizeof(float);

	/* float *dev_alpha; */
	/* int size_alpha = Nx_Ny * sizeof(float); */

	float *dev_f_minus_fo;
	int size_f_minus_fo = Nx_Ny * sizeof(float);

	// Allocation

	hipMalloc((void**) &dev_rr1, size_rr1);
	hipMalloc((void**) &dev_rr2, size_rr2);
	hipMalloc((void**) &dev_rr3, size_rr3);
	hipMalloc((void**) &dev_rr4, size_rr4);
	hipMalloc((void**) &dev_z, size_z);
	hipMalloc((void**) &dev_Lu, size_Lu);
	hipMalloc((void**) &dev_f, size_f);
	hipMalloc((void**) &dev_df, size_df);
	/* cudaMalloc((void**) &dev_alpha, size_alpha); */
	hipMalloc((void**) &dev_f_minus_fo, size_f_minus_fo);

	hipMemset(dev_rr1, 0.f, size_rr1);
	hipMemset(dev_rr2, 0.f, size_rr2);
	hipMemset(dev_rr3, 0.f, size_rr3);
	hipMemset(dev_rr4, 0.f, size_rr4);
	hipMemset(dev_f, 0.f, size_f);
	hipMemset(dev_Lu, 0.f, size_Lu);

	float *f = new float[Nx_Ny];
	float *f_minus_fo = new float[Nx_Ny];
	float epsilon = 0.f;

	for(int iter = 0; iter < I_MAX; iter++)
	{
		printf("\nIter: %d\n", iter);
		hipMemset(dev_u, 0.f, size_u);

		for(p = 0; p < 640; p += SENSOR_GROUP_SIZE)
		{
			int jp1 = jj[p];
			int jp2 = jj[p + SENSOR_GROUP_SIZE - 1];
			int ip1 = ii[p];
			int ip2 = ii[p + SENSOR_GROUP_SIZE - 1];

			if (jp2 < jp1)
			{
				int jp = jp1;
				jp1 = jp2;
				jp2 = jp;
			}

			if (ip2 < ip1)
			{
				int ip = ip1;
				ip1 = ip2;
				ip2 = ip;
			}

			// Boundary

			for(k = 1; k < NT - 1; k++)
			{
				propagation<<<Grid_Size, Block_Size>>>(jp1, jp2, ip1, ip2, dev_f, dev_u, k);
			}

			// Four corners

			propagation_at_corners<<<1, NT>>>(dev_u);
			difference_signal<<<NT - 2, 159>>>(dev_u, dev_g1, dev_g2, dev_g3, dev_g4, dev_rr1, dev_rr2, dev_rr3, dev_rr4, p);

			hipMemset(dev_z, 0.f, size_z);

			for(k = NT - 2; k > 0; k--)
			{
				backpropagation1<<<Grid_Size, Block_Size>>>(dev_z, dev_f, k);
				backpropagation2<<<1, NX>>>(dev_z, dev_rr1, dev_rr2, dev_rr3, dev_rr4, k);
			}

            laplace1<<<dim3(25, 25, 50), dim3(8, 8, 8)>>>(dev_u, dev_Lu);
            laplace2<<<1, 1>>>(dev_u, dev_Lu);

            hipMemset(dev_df, 0.f, size_df);

            update_differential<<<dim3(25, 25, 50), dim3(8, 8, 8)>>>(dev_df, dev_z, dev_Lu, dev_f);

			update_field<<<Grid_Size, Block_Size>>>(dev_f, dev_df, dev_f_minus_fo, dev_fo);
		}

		hipMemcpy(f_minus_fo, dev_f_minus_fo, size_f_minus_fo, hipMemcpyDeviceToHost);

		epsilon = norm(f_minus_fo, NX, NY) / norm(fo, NX, NY) * 100.f;

		printf("epsilon = %f\n", epsilon);

		if (epsilon < 20.f)
		{
			break;
		}
	}

	hipMemcpy(f, dev_f, size_f, hipMemcpyDeviceToHost);

	IO_Files(x, y, fo, f);

	// Free Variables

	hipFree(dev_x);
	hipFree(dev_y);
	hipFree(dev_fo);
	hipFree(dev_u);
	hipFree(dev_g1);
	hipFree(dev_g2);
	hipFree(dev_g3);
	hipFree(dev_g4);
	hipFree(dev_rr1);
	hipFree(dev_rr2);
	hipFree(dev_rr3);
	hipFree(dev_rr4);
	hipFree(dev_z);
	hipFree(dev_Lu);
	hipFree(dev_f);
	hipFree(dev_df);
	/* cudaFree(dev_alpha); */
	hipFree(dev_f_minus_fo);

	///////////////////////////

	/* float *image = new float[Nx_Ny]; */

	/* cudaMemcpy(image, dev_f, size_f, cudaMemcpyDeviceToHost); */

	/* ofstream file; */

	/* for(int yj = 0; yj < NY; yj++) */
	/* { */
	/* 	for(int xi = 0; xi < NX; xi++) */
	/* 	{ */
	/* 		file << image[xi + NX * yj]; */
	/* 		file << "\t"; */
	/* 	} */

	/* 	file << "\n"; */
	/* } */

	/* file.close(); */

	/////////////////////////
}

__global__ void field_setup(const float *x, const float *y, float *fo)
{
	// Map from threadIdx / BlockIdx to pixel position

	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;

	if((i < NX) && (j < NY))
	{
		int offset = i + PX * j;
		float value = 0.f;

		/* if(((sqrtf(powf(x[i] - 0.015f, 2.f) + powf(y[j] + 0.000f, 2.f))) <= 0.005f) || ((sqrtf(powf(x[i] + 0.015f, 2.f) + powf(y[j] + 0.000f, 2.f))) <= 0.005f)) */
		/* { */
		/* 	value = 0.06f; */
		/* } */
		/* else */
		/* { */
		/* 	if(sqrtf(x[i] * x[i] + y[j] * y[j]) <= 0.03f) */
		/* 	{ */
		/* 		value = 0.02f; */
		/* 	} */
		/* 	else */
		/* 	{ */
		/* 		value = 0.f; */
		/* 	} */
		/* } */

		float rc = 0.015f;
		float rp = 0.005f;
		/* float lim = 0.020f; */

		float sc = 0.03f;
		float sp = 0.05f;
		/* float sb = 0.02f; */

		if (powf(x[i], 2) + powf(y[j], 2) <= powf(rc, 2))
		{
			value = sc;
		}

		if (powf(x[i] - rc * cos(-30 * (3.14159265f / 180)), 2) + powf(y[j] - rc * sin(30 * (3.14159265f / 180)), 2) <= powf(rp, 2))
		{
			value = sp;
		}

		if (powf(x[i] + rc * cos(-30 * (3.14159265f / 180)), 2) + powf(y[j] - rc * sin(30 * (3.14159265f / 180)), 2) <= powf(rp, 2))
		{
			value = sp;
		}

		if (powf(x[i], 2) + powf(y[j] + rc, 2) <= powf(rp, 2))
		{
			value = sp;
		}

		fo[offset] = value;


		/*int offset = i + NX * j;
		float value = 0.f;

		if (((sqrtf(powf(x[i] - 0.05f, 2.f) + powf(y[j] + 0.000f, 2.f))) <= 0.005f) || ((sqrtf(powf(x[i] + 0.05f, 2.f) + powf(y[j] + 0.000f, 2.f))) <= 0.005f))
		{
			value = 0.06f;
		}
		else
		{
			if (sqrtf(x[i] * x[i] + y[j] * y[j]) <= 0.03f)
			{
				value = 0.02f;
			}
			else
			{
				if ((x[i] >= -0.05f) && (x[i] <= 0.05f) && (y[j] >= -0.06f) && (y[j] <= -0.045f))
				{
					value = 0.04f;
				}
				else
				{
					if ((x[i] >= -0.03f) && (x[i] <= 0.00f) && (y[j] <= 0.065f) && (y[j] >= (0.04f - 0.5f * x[i])))
					{
						value = 0.03f;
					}
					else
					{
						if ((x[i] >= 0.00f) && (x[i] <= 0.03f) && (y[j] <= 0.065f) && (y[j] >= (0.04f + 0.5f * x[i])))
						{
							value = 0.03f;
						}
						else
						{
							value = 0.f;
						}
					}
				}
			}
		}

		fo[offset] = value;
		v[offset] = 1500.f * sqrtf(1.f + value);
		r[offset] = v[offset] * dt / hx;
		r2[offset] = powf(r[offset], 2.f);
		s[offset] = 2.f - 4.f * r2[offset];
		*/
	}
}

__global__ void propagation(int jp1, int jp2, int ip1, int ip2, const float *f, float *u, int k)
{
	// Map from threadIdx / BlockIdx to pixel position

	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;

	if((i < NX) && (j < NY))
	{
	    int si = threadIdx.x + 1;
	    int sj = threadIdx.y + 1;

	    __shared__ float s_u[BLOCK_Y + 2][BLOCK_X + 2];

	    s_u[sj][si] = u[i + PX * j + PX * PY * k];

	    if (sj == 1) 
			{
	        s_u[sj - 1][si] = u[i + PX * (j - 1) + PX * PY * k];
	    }

      else if (sj == BLOCK_Y) 
			{
            s_u[sj + 1][si] = u[i + PX * (j + 1) + PX * PY * k];
      }

      if (si == 1) 
			{
            s_u[sj][si - 1] = u[(i - 1) + PX * j + PX * PY * k];
      }
      else if (si == BLOCK_X) 
			{
            s_u[sj][si + 1] = u[(i + 1) + PX * j + PX * PY * k];
      }

      __syncthreads();

			float v = 1500.f * sqrtf(1.f + f[i + PX * j]);
			float r = v * dt / hx;
  	  float s = 2.f - 4.f * r * r;
  	  float val = 0;

        // at sensor
			if (((j + 1) >= jp1) && ((j + 1) <= jp2) && ((i + 1) >= ip1) && ((i + 1) <= ip2) && ((k + 1) <= 24))
			{
        float t = k * dt - tt;
				val =
				    v * v * dt * dt *
				    cosf(omegac * t) *
				    expf(-(t * t) / (2.f * tao * tao)) +
				    r * r *
				    (s_u[sj][si + 1] +
				    s_u[sj][si - 1] +
				    s_u[sj - 1][si] + 
			    	s_u[sj + 1][si]) +
				    s * s_u[sj][si] -
					  u[i + PX * j + PX * PY * (k - 1)];
			}

        // not at sensor or boundary
			else if (i != 0 && j != 0 && i != (NX - 1) && j != (NY - 1))
			{
					val =
					    r * r *
					    (s_u[sj][si + 1] +
					    s_u[sj][si - 1] +
					    s_u[sj - 1][si] + 
					    s_u[sj + 1][si]) +
					    s * s_u[sj][si] -
					    u[i + PX * j + PX * PY * (k - 1)];
			}

        // left boundary
      else if ((i == 0) && (j > 0) && (j < (NY - 1)))
			{
					val =
					    (2.f - r * (r + 2.f)) * s_u[sj][si] +
					    2.f * r * (1.f + r) * s_u[sj][si + 1] -
					    r * r * s_u[sj][si + 2] +
					    (2.f * r - 1.f) * u[i + PX * j + PX * PY * (k - 1)] -
					    2.f * r * u[(i + 1) +  PX * j + PX * PY * (k - 1)];
			}

        // right boundary
      else if ((i == NX - 1) && (j > 0) && (j < (NY - 1)))
			{
					val =
					    (2.f - 2.f * r - r * r) * s_u[sj][si] +
					    2.f * r * (1.f + r) * s_u[sj][si - 1] -
			    		r * r * s_u[sj][si - 2] +
			    		(2.f * r - 1.f) * u[i + PX * j + PX * PY * (k - 1)] -
			    		2.f * r * u[(i - 1) + PX * j + PX * PY * (k - 1)];
			}

        // top boundary
       else if ((j == 0) && (i > 0) && (i < (NX - 1)))
			{
					val =
					    (2.f - 2.f * r - r * r) * s_u[sj][si] +
					    2.f * r * (1.f + r) * s_u[sj + 1][si] -
					    r * r * s_u[sj + 2][si] +
					    (2.f * r - 1.f) * u[i + PX * j + PX * PY * (k - 1)] -
					    2.f * r * u[i + PX * (j + 1) + PX * PY * (k - 1)];
			}	

        // bottom boundary
        else if ((j == NY - 1) && (i > 0) && (i < (NX - 1)))
				{
						val =
						    (2.f - 2.f * r - r * r) * s_u[sj][si] +
						    2.f * r * (1.f + r) * s_u[sj - 1][si] -
			   				r * r * s_u[sj - 2][si] +
			   			 	(2.f * r - 1.f) * u[i + PX * j + PX * PY * (k - 1)] -
			   				 2.f * r * u[i + PX * (j - 1) + PX * PY * (k - 1)];
				}
	     else {
            return;
        }

        u[i + PX * j + PX * PY * (k + 1)] = val;

	}
}

__global__ void propagation_at_corners(float *u)
{
	int k = threadIdx.x;
	int Nx_Ny = PX * PY;
	int Nx_Ny_k = Nx_Ny * k;

	u[Nx_Ny_k] =
	    1.f / 2.f * (u[PX + k] + u[1 + k]);

	u[(PX - 1) + Nx_Ny_k] =
	    1.f / 2.f * (u[(PX - 2) + Nx_Ny_k] + u[(PX - 1) + PX + Nx_Ny_k]);

	u[(PY - 1) * PX + Nx_Ny_k] =
	    1.f / 2.f * (u[(PY - 2) * PX + Nx_Ny_k] + u[1 +(PY - 1) * PX + Nx_Ny_k]);

	u[(PX - 1) + (PY - 1) * PX + Nx_Ny_k] =
	    1.f / 2.f * (u[(PX - 2) + (PY - 1) * PX + Nx_Ny_k] + u[(PX - 1) + (PY - 2) * PX + Nx_Ny_k]);

}

__global__ void initial_signal(const float *u, float *g1, float *g2, float *g3, float *g4, int p)
{
	int i = threadIdx.x + 21;
	int k = blockIdx.x + 2;

	int Nx_Ny_k = PX * PY * k;
	int i_k_Nx_Nx_Nt_p = i + PX * k + PX * NT * p;

	g1[i_k_Nx_Nx_Nt_p] = u[i + PX * 180 + Nx_Ny_k];
	g3[i_k_Nx_Nx_Nt_p] = u[i + PX * 20 + Nx_Ny_k];

	g2[i_k_Nx_Nx_Nt_p] = u[180 + PX * i + Nx_Ny_k];
	g4[i_k_Nx_Nx_Nt_p] = u[20 + PX * i + Nx_Ny_k];
}

__global__ void difference_signal(const float *u, const float *g1, const float *g2, const float *g3, const float *g4, float *rr1, float *rr2, float *rr3, float *rr4, int p)
{
	int i = threadIdx.x + 21;
	int k = blockIdx.x + 2;

	int Nx_Ny_k = PX * PY * k;
	int i_k_Nx_Nx_Nt_p = i + k * PX + PX * NT * p;
	int i_Nx_k = i + PX * k;

	rr1[i_Nx_k] = g1[i_k_Nx_Nx_Nt_p] - u[i + PX * 180 + Nx_Ny_k];
	rr3[i_Nx_k] = g3[i_k_Nx_Nx_Nt_p] - u[i + PX * 20 + Nx_Ny_k];

	rr2[i_Nx_k] = g2[i_k_Nx_Nx_Nt_p] - u[180 + PX * i + Nx_Ny_k];
	rr4[i_Nx_k] = g4[i_k_Nx_Nx_Nt_p] - u[20 + PX * i + Nx_Ny_k];
}

__global__ void backpropagation1(float *z, const float *f, int k)
{
	// Map from threadIdx / BlockIdx to pixel position

	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;

	if((i >= 1) && (i < (NX - 1)) && (j >= 1) && (j < (NY - 1)))
	{
		z[i + PX * j + PX * PY * k] =
		    1500.f * 1500.f * (dt * dt) *
		    ((1.f + f[i + PX * (j - 1)]) * z[i + PX * (j - 1) + PX * PY * (k + 1)] +
		     (1.f + f[i + PX * (j + 1)]) * z[i + PX * (j + 1) + PX * PY * (k + 1)] +
		     (1.f + f[(i - 1) + PX * j]) * z[(i - 1) + PX * j + PX * PY * (k + 1)] +
		     (1.f + f[(i + 1) + PX * j]) * z[(i + 1) + PX * j + PX * PY * (k + 1)] -
		     4.f * (1.f + f[i + PX * j]) *
		     z[i + PX * j + PX * PY * (k + 1)]) / (h * h) +
		    2.f * z[i + PX * j + PX * PY * (k + 1)] -
		    z[i + PX * j + PX * PY * (k + 2)];
	}
}

__global__ void backpropagation2(float *z, const float *rr1, const float *rr2, const float *rr3, const float *rr4, int k)
{
	int i = threadIdx.x;

	if((i >= 21) && (i < 180))
	{
		z[i + PX * 180 + PX * PY * k] =
		    z[i + PX * 179 + PX * PY * k] +
		    rr1[i + PX * k] * h * 1000.f;

		z[i + PX * 20 + PX * PY * k] =
		    z[i + PX * 21 + PX * PY * k] +
		    rr3[i + PX * k] * h * 1000.f;

		z[180 + PX * i + PX * PY * k] =
		    z[179 + PX * i + PX * PY * k] +
		    rr2[i + PX * k] * h * 1000.f;

		z[20 + PX * i + PX * PY * k] =
		    z[21 + PX * i + PX * PY * k]
		    + rr4[i + PX * k] * h * 1000.f;
	}

	if((i >= 1) && (i < (NX - 1)))
	{
		z[i + PX * PY * k] =
		    z[i + PX + PX * PY * k];

		z[i + PX * (NY - 1) + PX * PY * k] =
		    z[i + PX * (NY - 2) + PX * PY * k];

		z[PX * i + PX * PY * k] =
		    z[1 + PX * i + PX * PY * k];

		z[(NX - 1) + PX * i + PX * PY * k] =
		    z[(NX - 2) + PX * i + PX * PY * k];
	}

    else if(i == 0)
	{
		z[PX * PY * k] =
		    (z[1 + PX * PY * k] +
		     z[PX + PX * PY * k]) / 2.f;

		z[(NX - 1) + PX * PY * k] =
		    (z[(NX - 2) + PX * PY * k] +
		     z[(NX - 1) + PX + PX * PY * k]) / 2.f;

		z[PX * (NY - 1) + PX * PY * k] =
		    (z[1 + PX * (NY - 1) + PX * PY * k] +
		     z[PX * (NY - 2) + PX * PY * k]) / 2.f;

		z[(NX - 1) + PX * (NY - 1) + PX * PY * k] =
		    (z[(NX - 2) + PX * (NY - 1) + PX * PY * k] +
		     z[(NX - 1) + PX * (NY - 2) + PX * PY * k]) / 2.f;
	}
}

__global__ void laplace1(const float *u, float *Lu)
{
	// Map from threadIdx / BlockIdx to pixel position

	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;
	int k = threadIdx.z + blockIdx.z * blockDim.z;

	if(i < (NX - 1) && j < (NY - 1) && (k + 1) < NT)
	{
	    if (i >= 1) {
            if (j >= 1) {
                Lu[i + PX * j + PX * PY * (k + 1)] =
                    (u[i + PX * (j - 1) + PX * PY * (k + 1)] +
                    u[i + PX * (j + 1) + PX * PY * (k + 1)] +
                    u[(i - 1) + PX * j + PX * PY * (k + 1)] +
                    u[(i + 1) + PX * j + PX * PY * (k + 1)] -
                    4.f * u[i + PX * j + PX * PY * (k + 1)]) / (h * h);
            }

            else {
                Lu[i + PX * PY * (k + 1)] =
                    (u[i + PX * PY * (k + 1)] +
                    u[i + PX + PX * PY * (k + 1)] +
                    u[(i - 1) + PX * PY * (k + 1)] +
                    u[(i + 1) + PX * PY * (k + 1)] -
                    4.f * u[i + PX * PY * (k + 1)]) / (h * h);

                Lu[i + PX * (NY - 1) + PX * PY * (k + 1)] =
                    (u[i + PX * (NY - 1) + PX * PY * (k + 1)] +
                    u[i + PX * (NY - 2) + PX * PY * (k + 1)] +
                    u[(i - 1) + PX * (NY - 1) + PX * PY * (k + 1)] +
                    u[(i + 1) + PX * (NY - 1) + PX * PY * (k + 1)] -
                    4.f * u[i + PX * (NY - 1) + PX * PY * (k + 1)]) / (h * h);

                Lu[PX * i + PX * PY * (k + 1)] =
                    (u[PX * i + PX * PY * (k + 1)] +
                    u[1 + PX * i + PX * PY * (k + 1)] +
                    u[PX * (i - 1) + PX * PY * (k + 1)] +
                    u[PX * (i + 1) + PX * PY * (k + 1)] -
                    4.f * u[PX * i + PX * PY * (k + 1)]) / (h * h);

                Lu[(NX - 1) + PX * i + PX * PY * (k + 1)] =
                    (u[(NX - 1) + PX * i + PX * PY * (k + 1)] +
                    u[(NX - 2) + PX * i + PX * PY * (k + 1)] +
                    u[(NX - 1) + PX * (i - 1) + PX * PY * (k + 1)] +
                    u[(NX - 1) + PX * (i + 1) + PX * PY * (k + 1)] -
                    4.f * u[(NX - 1) + PX * i + PX * PY * (k + 1)]) / (h * h);
            }
        }
	}
}

__global__ void laplace2(const float *u, float *Lu)
{
    # pragma unroll
    for (int k = 1; k < NT; ++k) {
        Lu[PX * PY * k] =
            (Lu[1 + PX * PY * k] +
            Lu[PX + PX * PY * k]) / 2.f;

        Lu[(NX - 1) + PX * PY * k] =
            (Lu[(NX - 2) + PX * PY * k] +
            Lu[(NX - 1) + PX + PX * PY * k]) / 2.f;

        Lu[PX * (NY - 1) + PX * PY * k] =
            (Lu[1 + PX * (NY - 1) + PX * PY * k] +
            Lu[PX * (NY - 2) + PX * PY * k]) / 2.f;

        Lu[(NX - 1) + PX * (NY - 1) + PX * PY * k] =
            (Lu[(NX - 2) + PX * (NY - 1) + PX * PY * k] +
            Lu[(NX - 1) + PX * (NY - 2) + PX * PY * k]) / 2.f;
    }
}

__global__ void update_differential(float *df, const float *z, const float *Lu, const float *f)
{
	// Map from threadIdx / BlockIdx to pixel position

	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;
	int k = threadIdx.z + blockIdx.z * blockDim.z;

	if(i < NX && j < NY && (k + 1) < NT) {
		/* df[i + NX * j] += z[i + NX * j + NX * NY * (k + 1)] * Lu[i + NX * j + NX * NY * (k + 1)] / (1.f + f[i + NX * j]); */

		atomicAdd(&df[i + PX * j],
		             z[i + PX * j + PX * PY * (k + 1)] *
		            Lu[i + PX * j + PX * PY * (k + 1)] /
		            (1.f + f[i + PX * j]));
	}
}

/* __global__ void update_field(float *alpha, float *f, const float *df, float *f_minus_fo, const float *fo) */
__global__ void update_field(float *f, const float *df, float *f_minus_fo, const float *fo)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;

	if((i < NX) && (j < NY))
	{
		int offset = i + PX * j;

        bool flag = (i >= 21) && (i < 180) && (j >= 21) && (j < 180);

        float alpha = flag ? 1.f : 0.f;

		f[offset] += 20000.f * alpha * df[offset];
		f_minus_fo[offset] = f[offset] - fo[offset];
	}
}

__global__ void reset(const float *f, float *v, float *r, float *r2, float *s)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;

	if((i < NX) && (j < NY))
	{
		int offset = i + PX * j;

		v[offset] = 1500.f * sqrtf(1.f + f[offset]);
		r[offset] = v[offset] * dt / hx;
		r2[offset] = r[offset] * r[offset];
		s[offset] = 2.f - 4.f * r2[offset];
	}
}

void IO_Files(float *x, float *y, float *fo, float *f)
{
	int i = 0, j = 0;
	/* int k = 0; */

	// I/O Files

	ofstream x_file, y_file;
	ofstream fo_file;
	ofstream f_file;

	x_file.open("dev_x.txt");
	y_file.open("dev_y.txt");
	fo_file.open("dev_f0.txt");
	f_file.open("dev_f.txt");

	for(i = 0; i < NX; i++)
	{
		x_file << x[i];
		x_file << "\n";
	}

	for(j = 0; j < NX; j++)
	{
		y_file << y[j];
		y_file << "\n";
	}

	for(j = 0; j < NY; j++)
	{
		for(i = 0; i < NX; i++)
		{
			fo_file << fo[i + PX * j];
			fo_file << " ";
		}

		fo_file << "\n";
	}

	for(j = 0; j < NY; j++)
	{
		for(i = 0; i < NX; i++)
		{
			f_file << f[i + PX * j];
			f_file << " ";
		}

		f_file << "\n";
	}

	x_file.close();
	y_file.close();
	fo_file.close();
	f_file.close();
}

float norm(float *A, int nx, int ny)
{
	float sum = 0;

	for (int j = 0; j < ny; ++j)
	    for (int i = 0; i < nx; ++i)
	        sum += A[i + PX * j] * A[i + PX * j];

	return sqrtf(sum);
}
