#include "hip/hip_runtime.h"
// HEADERS

#include <iostream>
#include <iomanip>
#include <string>
#include <limits>
#include <stdlib.h>
#include <fstream>
#include <math.h>
#include <time.h>

using namespace std;

// DEFINITIONS

#define NX 192				//was 201
#define NY 192				//was 201
#define NT 401

#define NS 640 							//number of sensors

#define BLOCK_X 16
#define BLOCK_Y 16

template <typename T, int Nx>
struct Get2d
{
	__device__
	Get2d(T* _ptr)
		: ptr(_ptr)
	{}

	__device__
	T& operator()(int i, int j)
	{
		return ptr[i + Nx * j];
	}

	__device__
	const T& operator()(int i, int j) const
	{
		return ptr[i + Nx * j];
	}

	T* ptr;
};

template <typename T, int Nx, int Ny>
struct Get3d
{
	__device__
	Get3d(T* _ptr)
		: ptr(_ptr)
	{}

	__device__
	T& operator()(int i, int j, int k)
	{
		return ptr[i + Nx * j + Nx * Ny * k];
	}

	__device__
	const T& operator()(int i, int j, int k) const
	{
		return ptr[i + Nx * j + Nx * Ny * k];
	}

	T* ptr;
};

template <typename T, int Nx, int Ny, int Nz>
struct Get4d
{
	__device__
	Get4d(T* _ptr)
		: ptr(_ptr)
	{}

	__device__
	T& operator()(int i, int j, int k, int l)
	{
		return ptr[i + Nx * j + Nx * Ny * k + Nx * Ny * Nz * l];
	}

	__device__
	const T& operator()(int i, int j, int k, int l) const
	{
		return ptr[i + Nx * j + Nx * Ny * k + Nx * Ny * Nz * l];
	}

	T* ptr;
};

__constant__ float hx = 0.001f;
__constant__ float hy = 0.001f; // pixel size
__constant__ float h = 0.001f;

/* __constant__ float T = 1.3333e-04f; // 0.2f / 1500.f; */
__constant__ float dt = 3.3333e-07f; // T / 400.f;
/* __constant__ float fre = 125000.f; */
__constant__ float omegac = 7.8540e+05f; // 2.f * pi * fre; // wavelength
__constant__ float tao = 4.0000e-06f; // pi / omegac;
__constant__ float tt = 8.1573e-06f; // sqrtf(6.f * logf(2.f)) * tao; // time  delay

// FUNCTIONS DECLARATION

void Ultrasonic_Tomography(int group_size, float target_epsilon, int max_iterations, int ti);
void Position_Transducers (int *&, int *&, int);
void IO_Files(float*, float*, float*, float*);
float norm(float*, int, int);

template <typename T>
__host__ __device__
T& get(T* ptr, int i, int j = 0, int k = 0, int nx = NX, int ny = NY)
{
	return ptr[i + nx * j + nx * ny * k];
}

__global__ void field_setup(const float *x, const float *y, Get2d<float, NX> fo)
{
	// Map from threadIdx / BlockIdx to pixel position

	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;

	if((i < NX) && (j < NY))
	{
		float value = 0.f;

		float rc = 0.015f;
		float rp = 0.005f;

		float sc = 0.03f;
		float sp = 0.05f;

		if (powf(x[i], 2) + powf(y[j], 2) <= powf(rc, 2))
		{
			value = sc;
		}

		if (powf(x[i] - rc * cos(-30 * (3.14159265f / 180)), 2) + powf(y[j] - rc * sin(30 * (3.14159265f / 180)), 2) <= powf(rp, 2))
		{
			value = sp;
		}

		if (powf(x[i] + rc * cos(-30 * (3.14159265f / 180)), 2) + powf(y[j] - rc * sin(30 * (3.14159265f / 180)), 2) <= powf(rp, 2))
		{
			value = sp;
		}

		if (powf(x[i], 2) + powf(y[j] + rc, 2) <= powf(rp, 2))
		{
			value = sp;
		}

		fo(i, j) = value;
	}
}

__global__ void propagation(
		int jp1, int jp2, int ip1, int ip2, 
		const Get2d<float, NX> f,
		Get3d<float, NX, NY> u, 
		int k)
{
	// Map from threadIdx / BlockIdx to pixel position

	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;

	if(i < NX && j < NY) {
		float v = 1500.f * sqrtf(1.f + f(i, j));
		float r = v * dt / hx;
		float s = 2.f - 4.f * r * r;

		float val; // will hold new u at (i, j, k + 1)

		// not at boundary
		if (i != 0 && i != NX - 1 && j != 0 && j != NY - 1) {

			val =
				r * r *
				(u(i+1, j, k) +
				 u(i-1, j, k) +
				 u(i, j-1, k) +
				 u(i, j+1, k)) +
				s * u(i, j, k) -
				u(i, j, k-1);

			// at sensor, k <= 24
			if (j + 1 >= jp1 && j + 1 <= jp2 && i + 1 >= ip1 && i + 1 <= ip2 && k + 1 <= 24) {
				float t = k * dt - tt;

				// add wave value
				val +=
					v * v * dt * dt *
					cosf(omegac * t) *
					expf(-(t * t) / (2.f * tao * tao));
			}
		}

		// at boundary
		else {

			// boundary booleans
			bool top = (j == 0);
			bool bottom = (j == NY - 1);
			bool left = (i == 0);
			bool right = (i == NX - 1);

			// index variables for different boundary cases
			int ja = top ? (j + 1) : bottom ? (j - 1) : j;
			int jb = top ? (j + 2) : bottom ? (j - 2) : j;

			int ia = left ? (i + 1) : right ? (i - 1) : i;
			int ib = left ? (i + 2) : right ? (i - 2) : i;

			val =
				(2.f - 2.f * r - r * r) * u(i, j, k) +
				2.f * r * (1.f + r) * u(ia, ja, k) -
				r * r * u(ib, jb, k) +
				(2.f * r - 1.f) * u(i, j, k-1) -
				2.f * r * u(ia, ja, k-1);
		}

		u(i, j, k+1) = val;

		/* if (k+1 == NT - 1) */
			/* printf("%e \t", val); */
	}
}

__global__ void propagation_at_corners(Get3d<float, NX, NY> u)
{
	int k = threadIdx.x + blockIdx.x * blockDim.x;

	if (k < NT) {
		u(0, 0, k) =
			1.f / 2.f * (u(0, 1, k) + u(1, 0, k));

		u(NX-1, 0, k) =
			1.f / 2.f * (u(NX-2, 0, k) + u(NX-1, 1, k));

		u(0, NY-1, k) =
			1.f / 2.f * (u(0, NY-2, k) + u(1, NY-1, k));

		u(NX-1, NY-1, k) =
			1.f / 2.f * (u(NX-2, NY-1, k) + u(NX-1, NY-2, k));
	}
}

__global__ void initial_signal(
		const Get3d<float, NX, NY> u,
		Get3d<float, NX, NT> g_bottom,
		Get3d<float, NX, NT> g_right,
		Get3d<float, NX, NT> g_top,
		Get3d<float, NX, NT> g_left,
		int p)
{
	int i = threadIdx.x;
	int k = blockIdx.x;

	// store values at bottom sensor row of u
	g_bottom(i+21, k+2, p) =
		u(i+21, 180, k+2);

	// store values at top sensor row of u
	g_top(i+21, k+2, p) =
		u(i+21, 20, k+2);



	// store values at right sensor column of u
	g_right(i+21, k+2, p) =
		u(180, i+21, k+2);


	// store values at left sensor column of u
	g_left(i+21, k+2, p) =
		u(20, i+21, k+2);

	/* printf("%e \t", u(20, i+21, k+2)); */
}

__global__ void difference_signal(
		const Get3d<float, NX, NY> u,
		const Get3d<float, NX, NT> g_bottom,
		const Get3d<float, NX, NT> g_right,
		const Get3d<float, NX, NT> g_top,
		const Get3d<float, NX, NT> g_left,
		Get2d<float, NX> rr_bottom,
		Get2d<float, NX> rr_right,
		Get2d<float, NX> rr_top,
		Get2d<float, NX> rr_left,
		int p)
{
	int i = threadIdx.x;
	int k = blockIdx.x;

	// store difference at time k+2 of original signal
	// and current signal at bottom sensor row
	rr_bottom(i+21, k+2) =
		g_bottom(i+21, k+2, p) -
		u(i+21, 180, k+2);

	/* printf("%e ", rr_bottom(i+21, k+2)); */

	// store difference at time k+2 of original signal
	// and current signal at top sensor row
	rr_top(i+21, k+2) =
		g_top(i+21, k+2, p) -
		u(i+21, 20, k+2);

	// store difference at time k+2 of original signal
	// and current signal at right sensor column
	rr_right(i+21, k+2) =
		g_right(i+21, k+2, p) -
		u(180, i+21, k+2);

	// store difference at time k+2 of original signal
	// and current signal at left sensor column
	rr_left(i+21, k+2) =
		g_left(i+21, k+2, p) -
		u(20, i+21, k+2);
}

__global__ void backpropagation1(
		Get3d<float, NX, NY> z,
		const Get2d<float, NX> f,
		int k)
{
	// Map from threadIdx / BlockIdx to pixel position

	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;

	if(i >= 1 && i < (NX - 1) && j >= 1 && j < (NY - 1))
	{
		z(i, j, k) =
			1500.f * 1500.f * (dt * dt) *
			((1.f + f(i, j-1)) * z(i, j-1, k+1) +
			 (1.f + f(i, j+1)) * z(i, j+1, k+1) +
			 (1.f + f(i-1, j)) * z(i-1, j, k+1) +
			 (1.f + f(i+1, j)) * z(i+1, j, k+1) -
			 4.f * (1.f + f(i, j)) *
			 z(i, j, k+1)) / (h * h) +
			2.f * z(i, j, k+1) -
			z(i, j, k+2);

		/* if (k == 1) */
			/* printf("%e \t", z(i, j, k)); */
	}
}

__global__ void backpropagation2(
		Get3d<float, NX, NY> z,
		const Get2d<float, NX> rr_bottom,
		const Get2d<float, NX> rr_right,
		const Get2d<float, NX> rr_top,
		const Get2d<float, NX> rr_left,
		int k)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;

	if(i >= 21 && i < 180) {
		z(i, 180, k) =
			z(i, 179, k) +
			rr_bottom(i, k) * h * 1000.f;

		z(i, 20, k) =
			z(i, 21, k) +
			rr_top(i, k) * h * 1000.f;

		z(180, i, k) =
			z(179, i, k) +
			rr_right(i, k) * h * 1000.f;

		z(20, i, k) =
			z(21, i, k) +
			rr_left(i, k) * h * 1000.f;
	}

	if (i >= 1 && i < (NX - 1)) {
		z(i, 0, k) =
			z(i, 1, k);

		z(i, NY-1, k) =
			z(i, NY-2, k);

		z(0, i, k) =
			z(1, i, k);

		z(NX-1, i, k) =
			z(NX-2, i, k);
	}

	else if (i == 0) {
		z(0, 0, k) =
			(z(1, 0, k) +
			 z(0, 1, k)) / 2.f;

		z(NX-1, 0, k) =
			(z(NX-2, 0, k) +
			 z(NX-1, 1, k)) / 2.f;

		z(0, NY-1, k) =
			(z(1, NY-1, k) +
			 z(0, NY-2, k)) / 2.f;

		z(NX-1, NY-1, k) =
			(z(NX-2, NY-1, k) +
			 z(NX-1, NY-2, k)) / 2.f;
	}
}

__global__ void laplace(
		const Get3d<float, NX, NY> u,
		Get3d<float, NX, NY> Lu)
{
	// Map from threadIdx / BlockIdx to pixel position

	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;
	int k = threadIdx.z + blockIdx.z * blockDim.z;

	if (i < NX && j < NY && (k + 1) < NT) {

		int j_prev = (j > 0) ? j - 1 : j;
		int j_next = (j < NY - 1) ? j + 1 : j;

		int i_prev = (i > 0) ? i - 1 : i;
		int i_next = (i < NX - 1) ? i + 1 : i;

		Lu(i, j, k+1) =
			(u(i, j_prev, k+1) +
			 u(i, j_next, k+1) +
			 u(i_prev, j, k+1) +
			 u(i_next, j, k+1) -
			 4.f * u(i, j, k+1)) / (h * h);
	}
}

__global__ void laplace_corners(const Get3d<float, NX, NY> u, Get3d<float, NX, NY> Lu)
{
	int k = threadIdx.x + blockIdx.x * blockDim.x;

	if ((k + 1) < NT) {
		Lu(0, 0, k+1) =
			(Lu(1, 0, k+1) +
			 Lu(0, 1, k+1)) / 2.f;

		Lu(NX-1, 0, k+1) =
			(Lu(NX-2, 0, k+1) +
			 Lu(NX-1, 1, k+1)) / 2.f;

		Lu(0, NY-1, k+1) =
			(Lu(1, NY-1, k+1) +
			 Lu(0, NY-2, k+1)) / 2.f;

		Lu(NX-1, NY-1, k+1) =
			(Lu(NX-2, NY-1, k+1) +
			 Lu(NX-1, NY-2, k+1)) / 2.f;
	}
}

__global__ void update_differential(
		Get2d<float, NX> df,
		Get3d<float, NX, NY> const z,
		Get3d<float, NX, NY> const Lu,
		Get2d<float, NX> const f)
{
	// Map from threadIdx / BlockIdx to pixel position

	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;
	int k = threadIdx.z + blockIdx.z * blockDim.z;

	if(i < NX && j < NY && (k + 1) < NT) {

		atomicAdd(
			&df(i, j),
			z(i, j, k+1) *
			Lu(i, j, k+1) /
			(1.f + f(i, j)));
	}
}

__global__ void update_field(
		Get2d<float, NX> f,
		Get2d<float, NX> const df,
		Get2d<float, NX> f_minus_fo,
		Get2d<float, NX> const fo)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;

	if (i < NX && j < NY)
	{
		bool in_sensor_field = (i >= 21) && (i < 180) && (j >= 21) && (j < 180);

		float alpha = in_sensor_field ? 1.f : 0.f;

		f(i, j) += 20000.f * alpha * df(i, j);
		f_minus_fo(i, j) = f(i, j) - fo(i, j);
	}
}


// MAIN PROGRAM

int main(int argc, char **argv)
{
	if (argc != 4) {
		cerr << "Usage: " << argv[0] << " <sensor group size> <target epsilon> <max iterations>\n\n";
		exit(1);
	}

	int group_size = stoi(argv[1]);
	float target_epsilon = stof(argv[2]);
	int max_iterations = stoi(argv[3]);

	if (max_iterations == -1)
		max_iterations = numeric_limits<int>::max();

	// Time measuring variables

	int ti = 0, tf = 0;

	// Function Execution

	// set floting-point precision on stdout and stderr
	cout << fixed << setprecision(10);
	cerr << fixed << setprecision(10);

	cerr << "Ultrasonic Tomography Running:\n\n";

	ti = clock();
	cerr << "ti = " << ti << "\n";

	Ultrasonic_Tomography(group_size, target_epsilon, max_iterations, ti);

	tf = clock();
	cerr << "tf = " << tf << "\n"
		 << "tt = " << tf - ti << "\n"
		 << "Total Seconds = " << (float)(tf - ti) / CLOCKS_PER_SEC << "\n";

	// End of the program

	return 0;
}

inline int grid_size(int n, int threads)
{
	return ceil(float(n) / threads);
}

// FUNCTIONS DEFINITION

void Ultrasonic_Tomography(int group_size, float target_epsilon, int max_iterations, int ti)
{
	// Simulation Variables

	float hx = 0.001f;
	float hy = 0.001f;

	int i = 0, j = 0, k = 0;

	float *x = new float[NX];
	float *y = new float[NY];
	float *fo = new float[NX * NY];


	dim3 Block_Size(BLOCK_X, BLOCK_Y);
	dim3 Grid_Size(grid_size(NX, BLOCK_X), grid_size(NY, BLOCK_Y));

	// Variables of allocation

	float *dev_x;
	int size_x = NX * sizeof(float);

	float *dev_y;
	int size_y = NY * sizeof(float);

	float *dev_fo;
	int size_fo = NX * NY * sizeof(float);

	float *dev_u;
	int size_u = NX * NY * NT * sizeof(float);


	float *dev_g_bottom;
	float *dev_g_right;
	float *dev_g_top;
	float *dev_g_left;
	int size_g = NX * NT * (NS / group_size) * sizeof(float);


	hipMalloc((void**) &dev_x, size_x);
	hipMalloc((void**) &dev_y, size_y);
	hipMalloc((void**) &dev_fo, size_fo);
	hipMalloc((void**) &dev_u, size_u);
	hipMalloc((void**) &dev_g_bottom, size_g);
	hipMalloc((void**) &dev_g_right, size_g);
	hipMalloc((void**) &dev_g_top, size_g);
	hipMalloc((void**) &dev_g_left, size_g);

	hipMemset(dev_u, 0.f, size_u);
	hipMemset(dev_g_bottom, 0.f, size_g);
	hipMemset(dev_g_right, 0.f, size_g);
	hipMemset(dev_g_top, 0.f, size_g);
	hipMemset(dev_g_left, 0.f, size_g);

	// Environment Initialization

	for(i = 0; i < NX; i++)
	{
		x[i] = -0.1f + i * hx;
	}

	for(j = 0; j < NY; j++)
	{
		y[j] = -0.1f + j * hy;
	}

	hipMemcpy(dev_x, x, size_x, hipMemcpyHostToDevice);
	hipMemcpy(dev_y, y, size_y, hipMemcpyHostToDevice);

	field_setup<<<Grid_Size, Block_Size>>>(dev_x, dev_y, dev_fo);

	hipMemcpy(fo, dev_fo, size_fo, hipMemcpyDeviceToHost);

	// Position of the transducers
	int *ii, *jj;
	Position_Transducers(ii, jj, NS);


	dim3 threads_propagation(NX, 1, 1);
	dim3 grid_propagation(
			grid_size(NX, threads_propagation.x),
			grid_size(NY, threads_propagation.y));

	int p;
	for(p = 0; p < NS; p += group_size)
	{
		hipMemset(dev_u, 0.f, size_u);

		int jp1 = jj[p];
		int jp2 = jj[p + group_size - 1];
		int ip1 = ii[p];
		int ip2 = ii[p + group_size - 1];

		if (jp2 < jp1)
		{
			int jp = jp1;
			jp1 = jp2;
			jp2 = jp;
		}

		if (ip2 < ip1)
		{
			int ip = ip1;
			ip1 = ip2;
			ip2 = ip;
		}

		// Boundary

		for(k = 1; k < NT - 1; k++)
		{
			propagation<<<grid_propagation, threads_propagation>>>(jp1, jp2, ip1, ip2, dev_fo, dev_u, k);
		}

		// Four corners

		propagation_at_corners<<<NT, 1>>>(dev_u);

		initial_signal<<<NT - 2, 159>>>(dev_u, dev_g_bottom, dev_g_right, dev_g_top, dev_g_left, p / group_size);
	}

	// Kaczmarz method
	// propagation

	float *dev_rr_bottom;
	int size_rr_bottom = NX * NT * sizeof(float);

	float *dev_rr_right;
	int size_rr_right = NX * NT * sizeof(float);

	float *dev_rr_top;
	int size_rr_top = NX * NT * sizeof(float);

	float *dev_rr_left;
	int size_rr_left = NX * NT * sizeof(float);

	float *dev_z;
	int size_z = NX * NY * (NT + 1) * sizeof(float);

	float *dev_Lu;
	int size_Lu = NX * NY * NT * sizeof(float);

	float *dev_f;
	int size_f = NX * NY * sizeof(float);

	float *dev_df;
	int size_df = NX * NY * sizeof(float);

	float *dev_f_minus_fo;
	int size_f_minus_fo = NX * NY * sizeof(float);

	// Allocation

	hipMalloc((void**) &dev_rr_bottom, size_rr_bottom);
	hipMalloc((void**) &dev_rr_right, size_rr_right);
	hipMalloc((void**) &dev_rr_top, size_rr_top);
	hipMalloc((void**) &dev_rr_left, size_rr_left);
	hipMalloc((void**) &dev_z, size_z);
	hipMalloc((void**) &dev_Lu, size_Lu);
	hipMalloc((void**) &dev_f, size_f);
	hipMalloc((void**) &dev_df, size_df);
	hipMalloc((void**) &dev_f_minus_fo, size_f_minus_fo);

	hipMemset(dev_rr_bottom, 0.f, size_rr_bottom);
	hipMemset(dev_rr_right, 0.f, size_rr_right);
	hipMemset(dev_rr_top, 0.f, size_rr_top);
	hipMemset(dev_rr_left, 0.f, size_rr_left);
	hipMemset(dev_f, 0.f, size_f);
	hipMemset(dev_Lu, 0.f, size_Lu);

	float *f = new float[NX * NY];
	float *f_minus_fo = new float[NX * NY];

	// initialize epsilon values
	float prev_epsilon = std::numeric_limits<float>::infinity();
	float curr_epsilon = -std::numeric_limits<float>::infinity();

	cerr << "writing convergence to 'art_convergence.txt'...\n"
		 << "writing time to 'art_time.txt'...\n";

	ofstream convergence_file("art_convergence.txt");
	ofstream time_file("art_time.txt");

	dim3 threads_backpropagation1(NX, 1, 1);
	dim3 grid_backpropagation1(
			grid_size(NX, threads_backpropagation1.x),
			grid_size(NY, threads_backpropagation1.y));

	dim3 threads_laplace(96, 2, 1);
	dim3 grid_laplace(
			grid_size(NX, threads_laplace.x),
			grid_size(NY, threads_laplace.y),
			grid_size(NT, threads_laplace.z));

	dim3 threads_differential(96, 2, 1);
	dim3 grid_differential(
			grid_size(NX, threads_differential.x),
			grid_size(NY, threads_differential.y),
			grid_size(NT, threads_differential.z));

	for(int iter = 0; iter < max_iterations; iter++)
	{
		cout << "\nIter: " << iter << "\n";
		hipMemset(dev_u, 0.f, size_u);

		for(p = 0; p < NS; p += group_size)
		{
			int jp1 = jj[p];
			int jp2 = jj[p + group_size - 1];
			int ip1 = ii[p];
			int ip2 = ii[p + group_size - 1];

			if (jp2 < jp1)
			{
				int jp = jp1;
				jp1 = jp2;
				jp2 = jp;
			}

			if (ip2 < ip1)
			{
				int ip = ip1;
				ip1 = ip2;
				ip2 = ip;
			}

			// Boundary

			for(k = 1; k < NT - 1; k++)
			{
				propagation<<<grid_propagation, threads_propagation>>>(jp1, jp2, ip1, ip2, dev_f, dev_u, k);
			}

			// Four corners

			propagation_at_corners<<<NT, 1>>>(dev_u);
			difference_signal<<<NT - 2, 159>>>(dev_u, dev_g_bottom, dev_g_right, dev_g_top, dev_g_left, dev_rr_bottom, dev_rr_right, dev_rr_top, dev_rr_left, p / group_size);

			hipMemset(dev_z, 0.f, size_z);

			for(k = NT - 2; k > 0; k--)
			{
				backpropagation1<<<grid_backpropagation1, threads_backpropagation1>>>(dev_z, dev_f, k);
				backpropagation2<<<NX, 1>>>(dev_z, dev_rr_bottom, dev_rr_right, dev_rr_top, dev_rr_left, k);
			}

			laplace<<<grid_laplace, threads_laplace>>>(dev_u, dev_Lu);
			laplace_corners<<<NT, 1>>>(dev_u, dev_Lu);

			hipMemset(dev_df, 0.f, size_df);
			update_differential<<<grid_differential, threads_differential>>>(dev_df, dev_z, dev_Lu, dev_f);

			update_field<<<Grid_Size, Block_Size>>>(dev_f, dev_df, dev_f_minus_fo, dev_fo);
		}

		hipMemcpy(f_minus_fo, dev_f_minus_fo, size_f_minus_fo, hipMemcpyDeviceToHost);

		curr_epsilon = norm(f_minus_fo, NX, NY) / norm(fo, NX, NY) * 100.f;
		float current_t = (float)(clock()-ti) / CLOCKS_PER_SEC;

		convergence_file << curr_epsilon << " ";
		time_file << current_t << " ";

		cout << "epsilon = " << curr_epsilon << "\n";

		// stop if reached target epsilon
		if (curr_epsilon <= target_epsilon) {
			break;
		}

		// stop if epsilon diverges
		if (curr_epsilon > prev_epsilon ||
				std::isnan(curr_epsilon)) {
			break;
		}

		// update prev_epsilon
		prev_epsilon = curr_epsilon;
	}

	cout << endl;

	hipMemcpy(f, dev_f, size_f, hipMemcpyDeviceToHost);

	IO_Files(x, y, fo, f);

	// Free Variables

	hipFree(dev_x);
	hipFree(dev_y);
	hipFree(dev_fo);
	hipFree(dev_u);
	hipFree(dev_g_bottom);
	hipFree(dev_g_right);
	hipFree(dev_g_top);
	hipFree(dev_g_left);
	hipFree(dev_rr_bottom);
	hipFree(dev_rr_right);
	hipFree(dev_rr_top);
	hipFree(dev_rr_left);
	hipFree(dev_z);
	hipFree(dev_Lu);
	hipFree(dev_f);
	hipFree(dev_df);
	hipFree(dev_f_minus_fo);

	delete [] x;
	delete [] y;
	delete [] fo;
	delete [] ii;
	delete [] jj;
	delete [] f;
	delete [] f_minus_fo;

	hipDeviceReset();
}

void Position_Transducers(int *&ii, int *&jj, int num)
{
//returns the (x,y) coordinates of the number of total transducers
	int p = 0;
	ii = (int*)malloc(num * sizeof(int));
	jj = (int*)malloc(num * sizeof(int));


	for(p = 0; p < 160; p++)
	{
		ii[p] = 21 + (p + 1);
		jj[p] = 181;
	}

	for(p = 160; p < 320; p++)
	{
		ii[p] = 181;
		jj[p] = 181 - ((p + 1) - 160);
	}

	for(p = 320; p < 480; p++)
	{
		ii[p] = 181 - ((p + 1) - 320);
		jj[p] = 21;
	}

	for(p = 480; p < num; p++)
	{
		ii[p] = 21;
		jj[p] = 21 + ((p + 1) - 480);
	}
}
void IO_Files(float *x, float *y, float *fo, float *f)
{
	int i = 0, j = 0;

	// I/O Files

	ofstream x_file, y_file;
	ofstream fo_file;
	ofstream f_file;

	cerr << "writing x to 'dev_x.txt'...\n"
		 << "writing y to 'dev_y.txt'...\n"
		 << "writing f0 to 'dev_f0.txt'...\n"
		 << "writing f to 'dev_f.txt'...\n\n";

	x_file.open("dev_x.txt");
	y_file.open("dev_y.txt");
	fo_file.open("dev_f0.txt");
	f_file.open("dev_f.txt");

	for(i = 0; i < NX; i++) {
		x_file << x[i];
		x_file << "\n";
	}

	for(j = 0; j < NX; j++) {
		y_file << y[j];
		y_file << "\n";
	}

	for(j = 0; j < NY; j++) {
		for(i = 0; i < NX; i++) {
			fo_file << get(fo, i, j);
			fo_file << " ";
		}

		fo_file << "\n";
	}

	for(j = 0; j < NY; j++) {
		for(i = 0; i < NX; i++) {
			f_file << get(f, i, j);
			f_file << " ";
		}

		f_file << "\n";
	}

	x_file.close();
	y_file.close();
	fo_file.close();
	f_file.close();
}

float norm(float *A, int nx, int ny)
{
	float sum = 0;

	for (int j = 0; j < ny; ++j)
		for (int i = 0; i < nx; ++i)
			sum += get(A, i, j) * get(A, i, j);

	return sqrtf(sum);
}
