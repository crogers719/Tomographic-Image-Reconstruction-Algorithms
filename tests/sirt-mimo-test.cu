#include "hip/hip_runtime.h"
// HEADERS

#include <iostream>
#include <iomanip>
#include <string>
#include <limits>
#include <stdlib.h>
#include <fstream>
#include <math.h>
#include <time.h>

using namespace std;

// DEFINITIONS

#define NX 192				//was 201
#define NY 192				//was 201
#define NT 401

#define NS 640 				//number of sensors

#define BLOCK_X 16
#define BLOCK_Y 16

__constant__ float hx = 0.001f;
__constant__ float hy = 0.001f; // pixel size
__constant__ float h = 0.001f;

/* __constant__ float T = 1.3333e-04f; // 0.2f / 1500.f; */
__constant__ float dt = 3.3333e-07f; // T / 400.f;
/* __constant__ float fre = 125000.f; */
__constant__ float omegac = 7.8540e+05f; // 2.f * pi * fre; // wavelength
__constant__ float tao = 4.0000e-06f; // pi / omegac;
__constant__ float tt = 8.1573e-06f; // sqrtf(6.f * logf(2.f)) * tao; // time delay

// FUNCTIONS DECLARATION

void Ultrasonic_Tomography(int group_size, float target_epsilon, int max_iterations, int ti);
void Position_Transducers (int *&, int *&, int);
void IO_Files(float*, float*, float*, float*);
float norm(float*, int, int);

__global__ void field_setup(const float*, const float*, float*);
__global__ void propagation(int, int, int, int, const float*, float*, int);
__global__ void propagation(int, int, int, int, const float*, float*, int, int);
__global__ void propagation_at_corners(float*);
__global__ void propagation_at_corners(float*, int);
__global__ void initial_signal(const float*, float*, float*, float*, float*, int);
__global__ void difference_signal(const float*, const float*, const float*, const float*, const float*, float*, float*, float*, float*, int);
__global__ void backpropagation1(float*, const float*, int, int);
__global__ void backpropagation2(float*, const float*, const float*, const float*, const float*, int, int);
__global__ void laplace(const float*, float*, int);
__global__ void laplace_corners(const float*, float*, int);
__global__ void update_differential(float*, const float*, const float*, const float*, int);
__global__ void print_differential(const float *df);
__global__ void update_field(float*, const float*, float*, const float*);

template <typename T>
__host__ __device__
T& get(T* ptr, int i, int j = 0, int k = 0, int nx = NX, int ny = NY)
{
	return ptr[i + nx * j + nx * ny * k];
}

template <typename T>
__host__ __device__
T& getn(T* ptr, int i, int j = 0, int k = 0, int l = 0, int nx = NX, int ny = NY, int nz = NT)
{
	return ptr[i + nx * j + nx * ny * k + nx * ny * nz * l];
}

__host__ __device__
inline int grid_size(int n, int threads)
{
	return ceil((float)(n) / threads);
}

__global__ void pre_sirt(
		const int group_size, const int num_groups,
		const int *ii, const int *jj,
		const float *fo, float *u,
		float *g_bottom, float *g_top,
		float *g_left, float *g_right
		)
{
	// unique id for each group
	int g = threadIdx.x + blockIdx.x * blockDim.x;

	if (g >= num_groups)
		return;

	int p = g * group_size;

	int jp1 = jj[p];
	int jp2 = jj[p + group_size - 1];
	int ip1 = ii[p];
	int ip2 = ii[p + group_size - 1];

	if (jp2 < jp1) {
		int jp = jp1;
		jp1 = jp2;
		jp2 = jp;
	}

	if (ip2 < ip1) {
		int ip = ip1;
		ip1 = ip2;
		ip2 = ip;
	}

	dim3 threads_propagation(NX, 1, 1);
	dim3 grid_propagation(
		grid_size(NX, threads_propagation.x),
		grid_size(NY, threads_propagation.y));

	for(int k = 1; k < NT - 1; k++)
		propagation<<<grid_propagation, threads_propagation>>>(jp1, jp2, ip1, ip2, fo, u, k, g);

	propagation_at_corners<<<NT, 1>>>(u, g);

	initial_signal<<<NT - 2, 159>>>(u, g_bottom, g_right, g_top, g_left, g);
}

__global__ void sirt(
		const int group_size, const int num_groups,
		const int *ii, const int *jj,
		float *f, const float *fo, float *u,
		const float *g_bottom, const float *g_top,
		const float *g_left, const float *g_right,
		float *rr_bottom, float *rr_top,
		float *rr_left, float *rr_right,
		float *z, float *Lu, float *df, float *f_minus_fo
		)
{
	// unique id for each group
	int g = threadIdx.x + blockIdx.x * blockDim.x;

	if (g >= num_groups)
		return;

	int p = g * group_size;

	int jp1 = jj[p];
	int jp2 = jj[p + group_size - 1];
	int ip1 = ii[p];
	int ip2 = ii[p + group_size - 1];

	if (jp2 < jp1) {
		int jp = jp1;
		jp1 = jp2;
		jp2 = jp;
	}

	if (ip2 < ip1) {
		int ip = ip1;
		ip1 = ip2;
		ip2 = ip;
	}

	dim3 threads_propagation(NX, 1, 1);
	dim3 grid_propagation(
		grid_size(NX, threads_propagation.x),
		grid_size(NY, threads_propagation.y));

	dim3 threads_backpropagation1(NX, 1, 1);
	dim3 grid_backpropagation1(
		grid_size(NX, threads_backpropagation1.x),
		grid_size(NY, threads_backpropagation1.y));

	dim3 threads_laplace(96, 2, 1);
	dim3 grid_laplace(
		grid_size(NX, threads_laplace.x),
		grid_size(NY, threads_laplace.y),
		grid_size(NT, threads_laplace.z));

	dim3 threads_differential(96, 2, 1);
	dim3 grid_differential(
		grid_size(NX, threads_differential.x),
		grid_size(NY, threads_differential.y),
		grid_size(NT, threads_differential.z));

	for (int k = 1; k < NT - 1; ++k)
		propagation<<<grid_propagation, threads_propagation>>>(jp1, jp2, ip1, ip2, f, u, k, g);

	propagation_at_corners<<<NT, 1>>>(u, g);
	difference_signal<<<NT - 2, 159>>>(u, g_bottom, g_right, g_top, g_left, rr_bottom, rr_right, rr_top, rr_left, g);

	for (int k = NT - 2; k > 0; --k) {
		backpropagation1<<<grid_backpropagation1, threads_backpropagation1>>>(z, f, k, g);
		backpropagation2<<<NX, 1>>>(z, rr_bottom, rr_right, rr_top, rr_left, k, g);
	}

	laplace<<<grid_laplace, threads_laplace>>>(u, Lu, g);
	laplace_corners<<<NT, 1>>>(u, Lu, g);

	update_differential<<<grid_differential, threads_differential>>>(df, z, Lu, f, g);
}


// MAIN PROGRAM

int main(int argc, char **argv)
{
	if (argc != 4) {
		cerr << "Usage: " << argv[0] << " <sensor group size> <target epsilon> <max iterations>\n\n";
		exit(1);
	}

	int group_size = stoi(argv[1]);
	float target_epsilon = stof(argv[2]);
	int max_iterations = stoi(argv[3]);

	if (max_iterations == -1)
		max_iterations = numeric_limits<int>::max();

	// Time measuring variables

	int ti = 0, tf = 0;

	// Function Execution

	// set floating-point precision on stdout and stderr
	cout << fixed << setprecision(10);
	cerr << fixed << setprecision(10);

	cout << "Ultrasonic Tomography Running:\n\n";

	ti = clock();
	cout << "ti = " << ti << "\n";

	Ultrasonic_Tomography(group_size, target_epsilon, max_iterations, ti);

	tf = clock();
	cout << "tf = " << tf << "\n"
		 << "tt = " << tf - ti << "\n"
		 << "Total Seconds = " << (float)(tf - ti) / CLOCKS_PER_SEC << "\n";

	// End of the program

	return 0;
}

// FUNCTIONS DEFINITION

void Ultrasonic_Tomography(int group_size, float target_epsilon, int max_iterations, int ti)
{

	// number of sensor groups that will be launched in parallel
	int num_groups = NS / group_size;

	// Simulation Variables

	float hx = 0.001f;
	float hy = 0.001f;

	int i = 0, j = 0;

	float *x = new float[NX];
	float *y = new float[NY];
	float *fo = new float[NX * NY];


	dim3 Block_Size(BLOCK_X, BLOCK_Y);
	dim3 Grid_Size(grid_size(NX, BLOCK_X), grid_size(NY, BLOCK_Y));

	// Variables of allocation

	float *dev_x;
	int size_x = NX * sizeof(float);

	float *dev_y;
	int size_y = NY * sizeof(float);

	float *dev_fo;
	int size_fo = NX * NY * sizeof(float);

	float *dev_u;
	int size_u = NX * NY * NT * num_groups * sizeof(float);


	float *dev_g_bottom;
	float *dev_g_right;
	float *dev_g_top;
	float *dev_g_left;
	int size_g = NX * NT * num_groups * sizeof(float);


	hipMalloc((void**) &dev_x, size_x);
	hipMalloc((void**) &dev_y, size_y);
	hipMalloc((void**) &dev_fo, size_fo);
	hipMalloc((void**) &dev_u, size_u);
	hipMalloc((void**) &dev_g_bottom, size_g);
	hipMalloc((void**) &dev_g_right, size_g);
	hipMalloc((void**) &dev_g_top, size_g);
	hipMalloc((void**) &dev_g_left, size_g);

	hipMemset(dev_u, 0.f, size_u);
	hipMemset(dev_g_bottom, 0.f, size_g);
	hipMemset(dev_g_right, 0.f, size_g);
	hipMemset(dev_g_top, 0.f, size_g);
	hipMemset(dev_g_left, 0.f, size_g);

	// Environment Initialization

	for(i = 0; i < NX; i++)
	{
		x[i] = -0.1f + i * hx;
	}

	for(j = 0; j < NY; j++)
	{
		y[j] = -0.1f + j * hy;
	}

	hipMemcpy(dev_x, x, size_x, hipMemcpyHostToDevice);
	hipMemcpy(dev_y, y, size_y, hipMemcpyHostToDevice);

	field_setup<<<Grid_Size, Block_Size>>>(dev_x, dev_y, dev_fo);

	hipMemcpy(fo, dev_fo, size_fo, hipMemcpyDeviceToHost);

	// Position of the transducers
	int *ii, *jj;
	Position_Transducers(ii, jj, NS);

	int *dev_ii;
	int *dev_jj;

	hipMalloc((void**) &dev_ii, NS * sizeof(int));
	hipMalloc((void**) &dev_jj, NS * sizeof(int));
	hipMemcpy(dev_ii, ii, NS * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_jj, jj, NS * sizeof(int), hipMemcpyHostToDevice);

	dim3 threads_propagation(NX, 1, 1);
	dim3 grid_propagation(
		grid_size(NX, threads_propagation.x),
		grid_size(NY, threads_propagation.y));

	hipMemset(dev_u, 0.f, size_u);

	pre_sirt<<<1, num_groups>>>(
		group_size, num_groups,
		dev_ii, dev_jj,
		dev_fo, dev_u,
		dev_g_bottom, dev_g_top,
		dev_g_left, dev_g_right);

	// Kaczmarz method
	// propagation

	float *dev_rr_bottom;
	float *dev_rr_right;
	float *dev_rr_top;
	float *dev_rr_left;
	int size_rr = NX * NT * num_groups * sizeof(float);

	float *dev_z;
	int size_z = NX * NY * (NT + 1) * num_groups * sizeof(float);

	float *dev_Lu;
	int size_Lu = NX * NY * NT * num_groups * sizeof(float);

	float *dev_f;
	int size_f = NX * NY * sizeof(float);

	float *dev_df;
	int size_df = NX * NY * sizeof(float);

	float *dev_f_minus_fo;
	int size_f_minus_fo = NX * NY * sizeof(float);

	// Allocation

	hipMalloc((void**) &dev_rr_bottom, size_rr);
	hipMalloc((void**) &dev_rr_right, size_rr);
	hipMalloc((void**) &dev_rr_top, size_rr);
	hipMalloc((void**) &dev_rr_left, size_rr);
	hipMalloc((void**) &dev_z, size_z);
	hipMalloc((void**) &dev_Lu, size_Lu);
	hipMalloc((void**) &dev_f, size_f);
	hipMalloc((void**) &dev_df, size_df);
	hipMalloc((void**) &dev_f_minus_fo, size_f_minus_fo);

	hipMemset(dev_rr_bottom, 0.f, size_rr);
	hipMemset(dev_rr_right, 0.f, size_rr);
	hipMemset(dev_rr_top, 0.f, size_rr);
	hipMemset(dev_rr_left, 0.f, size_rr);
	hipMemset(dev_f, 0.f, size_f);
	hipMemset(dev_Lu, 0.f, size_Lu);

	float *f = new float[NX * NY];
	float *f_minus_fo = new float[NX * NY];

	// initialize epsilon values
	float prev_epsilon = std::numeric_limits<float>::infinity();
	float curr_epsilon = -std::numeric_limits<float>::infinity();

	cerr << "writing convergence to 'sirt_convergence.txt'...\n"
		 << "writing time to 'sirt_time.txt'...\n";

	ofstream convergence_file("sirt_convergence.txt");
	ofstream time_file("sirt_time.txt");

	for(int iter = 0; iter < max_iterations; iter++)
	{
		cout << "\nIter: " << iter << "\n";
		hipMemset(dev_u, 0.f, size_u);
		hipMemset(dev_z, 0.f, size_z);
		hipMemset(dev_df, 0.f, size_df);

		sirt<<<1, num_groups>>>
			(group_size, num_groups,
			 dev_ii, dev_jj,
			 dev_f, dev_fo, dev_u,
			 dev_g_bottom, dev_g_top, dev_g_left, dev_g_right,
			 dev_rr_bottom, dev_rr_top, dev_rr_left, dev_rr_right,
			 dev_z, dev_Lu, dev_df, dev_f_minus_fo);

		/* print_differential<<<1, 1>>>(dev_df); */

		update_field<<<Grid_Size, Block_Size>>>(dev_f, dev_df, dev_f_minus_fo, dev_fo);

		hipMemcpy(f_minus_fo, dev_f_minus_fo, size_f_minus_fo, hipMemcpyDeviceToHost);

		curr_epsilon = norm(f_minus_fo, NX, NY) / norm(fo, NX, NY) * 100.f;
		float current_t = (float)(clock()-ti) / CLOCKS_PER_SEC;

		convergence_file << curr_epsilon << " ";
		time_file << current_t << " ";

		cout << "epsilon = " << curr_epsilon << "\n";

		// stop if reached target epsilon
		if (curr_epsilon <= target_epsilon) {
			break;
		}

		// stop if epsilon diverges
		if (curr_epsilon > prev_epsilon ||
				std::isnan(curr_epsilon)) {
			break;
		}

		// update prev_epsilon
		prev_epsilon = curr_epsilon;
	}

	cout << endl;

	hipError_t error = hipGetLastError();

	if (error != hipSuccess) {
		cerr << hipGetErrorString(error) << endl;
	}

	hipMemcpy(f, dev_f, size_f, hipMemcpyDeviceToHost);

	IO_Files(x, y, fo, f);

	// Free Variables

	size_t free, total;
	hipMemGetInfo(&free, &total);

	cerr << fixed << setprecision(4);

	cerr << "used mem:  " << float(total - free) / (1024 * 1024) << " MB\n"
		 << "free mem:  " << float(free) / (1024 * 1024)  << " MB\n"
		 << "total mem: " << float(total) / (1024 * 1024) << " MB\n\n";

	hipFree(dev_x);
	hipFree(dev_y);
	hipFree(dev_fo);
	hipFree(dev_u);
	hipFree(dev_g_bottom);
	hipFree(dev_g_right);
	hipFree(dev_g_top);
	hipFree(dev_g_left);
	hipFree(dev_rr_bottom);
	hipFree(dev_rr_right);
	hipFree(dev_rr_top);
	hipFree(dev_rr_left);
	hipFree(dev_z);
	hipFree(dev_Lu);
	hipFree(dev_f);
	hipFree(dev_df);
	hipFree(dev_f_minus_fo);

	delete [] x;
	delete [] y;
	delete [] fo;
	delete [] ii;
	delete [] jj;
	delete [] f;
	delete [] f_minus_fo;

	hipDeviceReset();
}

void Position_Transducers(int *&ii, int *&jj, int num)
{
//returns the (x,y) coordinates of the number of total transducers
	int p = 0;
	ii = (int*)malloc(num * sizeof(int));
	jj = (int*)malloc(num * sizeof(int));


	for(p = 0; p < 160; p++)
	{
		ii[p] = 21 + (p + 1);
		jj[p] = 181;
	}

	for(p = 160; p < 320; p++)
	{
		ii[p] = 181;
		jj[p] = 181 - ((p + 1) - 160);
	}

	for(p = 320; p < 480; p++)
	{
		ii[p] = 181 - ((p + 1) - 320);
		jj[p] = 21;
	}

	for(p = 480; p < num; p++)
	{
		ii[p] = 21;
		jj[p] = 21 + ((p + 1) - 480);
	}
}

__global__ void field_setup(const float *x, const float *y, float *fo)
{
	// Map from threadIdx / BlockIdx to pixel position

	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;

	if((i < NX) && (j < NY))
	{
		/* int offset = i + NX * j; */
		float value = 0.f;

		/* if(((sqrtf(powf(x[i] - 0.015f, 2.f) + powf(y[j] + 0.000f, 2.f))) <= 0.005f) || ((sqrtf(powf(x[i] + 0.015f, 2.f) + powf(y[j] + 0.000f, 2.f))) <= 0.005f)) */
		/* { */
		/*	value = 0.06f; */
		/* } */
		/* else */
		/* { */
		/*	if(sqrtf(x[i] * x[i] + y[j] * y[j]) <= 0.03f) */
		/*	{ */
		/*		value = 0.02f; */
		/*	} */
		/*	else */
		/*	{ */
		/*		value = 0.f; */
		/*	} */
		/* } */

		float rc = 0.015f;
		float rp = 0.005f;
		/* float lim = 0.020f; */

		float sc = 0.03f;
		float sp = 0.05f;
		/* float sb = 0.02f; */

		if (powf(x[i], 2) + powf(y[j], 2) <= powf(rc, 2))
		{
			value = sc;
		}

		if (powf(x[i] - rc * cos(-30 * (3.14159265f / 180)), 2) + powf(y[j] - rc * sin(30 * (3.14159265f / 180)), 2) <= powf(rp, 2))
		{
			value = sp;
		}

		if (powf(x[i] + rc * cos(-30 * (3.14159265f / 180)), 2) + powf(y[j] - rc * sin(30 * (3.14159265f / 180)), 2) <= powf(rp, 2))
		{
			value = sp;
		}

		if (powf(x[i], 2) + powf(y[j] + rc, 2) <= powf(rp, 2))
		{
			value = sp;
		}

		getn(fo, i, j) = value;
		/* fo(i, j) = value; */


		/*int offset = i + NX * j;
		float value = 0.f;

		if (((sqrtf(powf(x[i] - 0.05f, 2.f) + powf(y[j] + 0.000f, 2.f))) <= 0.005f) || ((sqrtf(powf(x[i] + 0.05f, 2.f) + powf(y[j] + 0.000f, 2.f))) <= 0.005f))
		{
			value = 0.06f;
		}
		else
		{
			if (sqrtf(x[i] * x[i] + y[j] * y[j]) <= 0.03f)
			{
				value = 0.02f;
			}
			else
			{
				if ((x[i] >= -0.05f) && (x[i] <= 0.05f) && (y[j] >= -0.06f) && (y[j] <= -0.045f))
				{
					value = 0.04f;
				}
				else
				{
					if ((x[i] >= -0.03f) && (x[i] <= 0.00f) && (y[j] <= 0.065f) && (y[j] >= (0.04f - 0.5f * x[i])))
					{
						value = 0.03f;
					}
					else
					{
						if ((x[i] >= 0.00f) && (x[i] <= 0.03f) && (y[j] <= 0.065f) && (y[j] >= (0.04f + 0.5f * x[i])))
						{
							value = 0.03f;
						}
						else
						{
							value = 0.f;
						}
					}
				}
			}
		}

		fo[offset] = value;
		v[offset] = 1500.f * sqrtf(1.f + value);
		r[offset] = v[offset] * dt / hx;
		r2[offset] = powf(r[offset], 2.f);
		s[offset] = 2.f - 4.f * r2[offset];
		*/
	}
}

/* __global__ void propagation(int jp1, int jp2, int ip1, int ip2, const float *f, float *u, int k) */
/* { */
/* 	// Map from threadIdx / BlockIdx to pixel position */

/* 	int i = threadIdx.x + blockIdx.x * blockDim.x; */
/* 	int j = threadIdx.y + blockIdx.y * blockDim.y; */

/* 	if(i < NX && j < NY) { */
/* 		float v = 1500.f * sqrtf(1.f + get(f, i, j)); */
/* 		float r = v * dt / hx; */
/* 		float s = 2.f - 4.f * r * r; */

/* 		float val; // will hold new u at (i, j, k + 1) */

/* 		// not at boundary */
/* 		if (i != 0 && i != NX - 1 && j != 0 && j != NY - 1) { */

/* 			val = */
/* 				r * r * */
/* 				(get(u, i+1, j, k) + */
/* 				 get(u, i-1, j, k) + */
/* 				 get(u, i, j-1, k) + */
/* 				 get(u, i, j+1, k)) + */
/* 				s * get(u, i, j, k) - */
/* 				get(u, i, j, k-1); */

/* 			// at sensor, k <= 24 */
/* 			if (j + 1 >= jp1 && j + 1 <= jp2 && i + 1 >= ip1 && i + 1 <= ip2 && k + 1 <= 24) { */
/* 				float t = k * dt - tt; */

/* 				// add wave value */
/* 				val += */
/* 					v * v * dt * dt * */
/* 					cosf(omegac * t) * */
/* 					expf(-(t * t) / (2.f * tao * tao)); */
/* 			} */
/* 		} */

/* 		// at boundary */
/* 		else { */

/* 			// index variables for different boundary cases */
/* 			// TODO: need better names */
/* 			int i_A, i_B, j_A, j_B; */

/* 			// top boundary */
/* 			if (j == 0) */
/* 			{ */
/* 				i_A = i; */
/* 				i_B = i; */
/* 				j_A = j + 1; */
/* 				j_B = j + 2; */
/* 			} */

/* 			// bottom boundary */
/* 			else if (j == NY - 1) */
/* 			{ */
/* 				i_A = i; */
/* 				i_B = i; */
/* 				j_A = j - 1; */
/* 				j_B = j - 2; */
/* 			} */

/* 			// left boundary */
/* 			else if (i == 0) */
/* 			{ */
/* 				i_A = i + 1; */
/* 				i_B = i + 2; */
/* 				j_A = j; */
/* 				j_B = j; */
/* 			} */

/* 			// right boundary */
/* 			else */
/* 			{ */
/* 				i_A = i - 1; */
/* 				i_B = i - 2; */
/* 				j_A = j; */
/* 				j_B = j; */
/* 			} */

/* 			val = */
/* 				(2.f - 2.f * r - r * r) * get(u, i, j, k) + */
/* 				2.f * r * (1.f + r) * get(u, i_A, j_A, k) - */
/* 				r * r * get(u, i_B, j_B, k) + */
/* 				(2.f * r - 1.f) * get(u, i, j, k-1) - */
/* 				2.f * r * get(u, i_A, j_A, k-1); */
/* 		} */

/* 		get(u, i, j, k+1) = val; */
/* 	} */
/* } */

__global__ void propagation(int jp1, int jp2, int ip1, int ip2, const float *f, float *u, int k, int g)
{
	// Map from threadIdx / BlockIdx to pixel position

	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;

	if(i < NX && j < NY) {
		float v = 1500.f * sqrtf(1.f + getn(f, i, j));
		float r = v * dt / hx;
		float s = 2.f - 4.f * r * r;

		float val; // will hold new u at (i, j, k + 1)

		// not at boundary
		if (i != 0 && i != NX - 1 && j != 0 && j != NY - 1) {

			val =
				r * r *
				(getn(u, i+1, j, k, g) +
				 getn(u, i-1, j, k, g) +
				 getn(u, i, j-1, k, g) +
				 getn(u, i, j+1, k, g)) +
				s * getn(u, i, j, k, g) -
				getn(u, i, j, k-1, g);

			// at sensor, k <= 24
			if (j + 1 >= jp1 && j + 1 <= jp2 && i + 1 >= ip1 && i + 1 <= ip2 && k + 1 <= 24) {
				float t = k * dt - tt;

				// add wave value
				val +=
					v * v * dt * dt *
					cosf(omegac * t) *
					expf(-(t * t) / (2.f * tao * tao));
			}
		}

		// at boundary
		else {

			// index variables for different boundary cases
			// TODO: need better names
			int i_A, i_B, j_A, j_B;

			// top boundary
			if (j == 0)
			{
				i_A = i;
				i_B = i;
				j_A = j + 1;
				j_B = j + 2;
			}

			// bottom boundary
			else if (j == NY - 1)
			{
				i_A = i;
				i_B = i;
				j_A = j - 1;
				j_B = j - 2;
			}

			// left boundary
			else if (i == 0)
			{
				i_A = i + 1;
				i_B = i + 2;
				j_A = j;
				j_B = j;
			}

			// right boundary
			else
			{
				i_A = i - 1;
				i_B = i - 2;
				j_A = j;
				j_B = j;
			}

			val =
				(2.f - 2.f * r - r * r) * getn(u, i, j, k, g) +
				2.f * r * (1.f + r) * getn(u, i_A, j_A, k, g) -
				r * r * getn(u, i_B, j_B, k, g) +
				(2.f * r - 1.f) * getn(u, i, j, k-1, g) -
				2.f * r * getn(u, i_A, j_A, k-1, g);
		}

		getn(u, i, j, k+1, g) = val;

		/* if (k+1 == NT - 1) */
			/* printf("%e \t", getn(u, i, j, k+1, g)); */
	}
}

/* __global__ void propagation_at_corners(float *u) */
/* { */
/* 	int k = threadIdx.x + blockIdx.x * blockDim.x; */

/* 	if (k < NT) { */
/* 		get(u, 0, 0, k) = */
/* 			1.f / 2.f * (get(u, 0, 1, k) + get(u, 1, 0, k)); */

/* 		get(u, NX-1, 0, k) = */
/* 			1.f / 2.f * (get(u, NX-2, 0, k) + get(u, NX-1, 1, k)); */

/* 		get(u, 0, NY-1, k) = */
/* 			1.f / 2.f * (get(u, 0, NY-2, k) + get(u, 1, NY-1, k)); */

/* 		get(u, NX-1, NY-1, k) = */
/* 			1.f / 2.f * (get(u, NX-2, NY-1, k) + get(u, NX-1, NY-2, k)); */
/* 	} */
/* } */

__global__ void propagation_at_corners(float *u, int g)
{
	int k = threadIdx.x + blockIdx.x * blockDim.x;

	if (k < NT) {
		getn(u, 0, 0, k, g) =
			1.f / 2.f * (getn(u, 0, 1, k, g) + getn(u, 1, 0, k, g));

		getn(u, NX-1, 0, k, g) =
			1.f / 2.f * (getn(u, NX-2, 0, k, g) + getn(u, NX-1, 1, k, g));

		getn(u, 0, NY-1, k, g) =
			1.f / 2.f * (getn(u, 0, NY-2, k, g) + getn(u, 1, NY-1, k, g));

		getn(u, NX-1, NY-1, k, g) =
			1.f / 2.f * (getn(u, NX-2, NY-1, k, g) + getn(u, NX-1, NY-2, k, g));
	}
}

__global__ void initial_signal(const float *u, float *g_bottom, float *g_right, float *g_top, float *g_left, int g)
{
	int i = threadIdx.x;
	int k = blockIdx.x;

	// store values at bottom sensor row of u
	getn(g_bottom, i+21, k+2, g, 0, NX, NT) =
		getn(u, i+21, 180, k+2, g);


	// store values at top sensor row of u
	getn(g_top, i+21, k+2, g, 0, NX, NT) =
		getn(u, i+21, 20, k+2, g);


	// store values at right sensor column of u
	getn(g_right, i+21, k+2, g, 0, NX, NT) =
		getn(u, 180, i+21, k+2, g);

	// store values at left sensor column of u
	getn(g_left, i+21, k+2, g, 0, NX, NT) =
		getn(u, 20, i+21, k+2, g);

	/* printf("%e \t", getn(u, 20, i+21, k+2, g)); */
}

__global__ void difference_signal(const float *u, const float *g_bottom, const float *g_right, const float *g_top, const float *g_left, float *rr_bottom, float *rr_right, float *rr_top, float *rr_left, int g)
{
	int i = threadIdx.x;
	int k = blockIdx.x;

	// store difference at time k+2 of original signal
	// and current signal at bottom sensor row
	getn(rr_bottom, i+21, k+2, g, 0, NX, NT) =
		getn(g_bottom, i+21, k+2, g, 0, NX, NT) -
		getn(u, i+21, 180, k+2, g);

	/* printf("%e \t", getn(rr_bottom, i+21, k+2, g, 0, NX, NT)); */

	// store difference at time k+2 of original signal
	// and current signal at top sensor row
	getn(rr_top, i+21, k+2, g, 0, NX, NT) =
		getn(g_top, i+21, k+2, g, 0, NX, NT) -
		getn(u, i+21, 20, k+2, g);

	// store difference at time k+2 of original signal
	// and current signal at right sensor column
	getn(rr_right, i+21, k+2, g, 0, NX, NT) =
		getn(g_right, i+21, k+2, g, 0, NX, NT) -
		getn(u, 180, i+21, k+2, g);

	// store difference at time k+2 of original signal
	// and current signal at left sensor column
	getn(rr_left, i+21, k+2, g, 0, NX, NT) =
		getn(g_left, i+21, k+2, g, 0, NX, NT) -
		getn(u, 20, i+21, k+2, g);
}

__global__ void backpropagation1(float *z, const float *f, int k, int g)
{
	// Map from threadIdx / BlockIdx to pixel position

	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;

	if(i >= 1 && i < (NX - 1) && j >= 1 && j < (NY - 1))
	{
		getn(z, i, j, k, g, NX, NY, NT+1) =
			1500.f * 1500.f * (dt * dt) *
			((1.f + getn(f, i, j-1)) * getn(z, i, j-1, k+1, g, NX, NY, NT+1) +
			 (1.f + getn(f, i, j+1)) * getn(z, i, j+1, k+1, g, NX, NY, NT+1) +
			 (1.f + getn(f, i-1, j)) * getn(z, i-1, j, k+1, g, NX, NY, NT+1) +
			 (1.f + getn(f, i+1, j)) * getn(z, i+1, j, k+1, g, NX, NY, NT+1) -
			 4.f * (1.f + getn(f, i, j)) *
			 getn(z, i, j, k+1, g, NX, NY, NT+1)) / (h * h) +
			2.f * getn(z, i, j, k+1, g, NX, NY, NT+1) -
			getn(z, i, j, k+2, g, NX, NY, NT+1);

		/* if (k == 1) */
			/* printf("%e \t", getn(z, i, j, k, g, NX, NY, NT+1)); */
	}
}

__global__ void backpropagation2(float *z, const float *rr_bottom, const float *rr_right, const float *rr_top, const float *rr_left, int k, int g)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;

	if(i >= 21 && i < 180) {
		getn(z, i, 180, k, g, NX, NY, NT+1) =
			getn(z, i, 179, k, g, NX, NY, NT+1) +
			getn(rr_bottom, i, k) * h * 1000.f;

		getn(z, i, 20, k, g, NX, NY, NT+1) =
			getn(z, i, 21, k, g, NX, NY, NT+1) +
			getn(rr_top, i, k, g, 0, NX, NT) * h * 1000.f;

		getn(z, 180, i, k, g, NX, NY, NT+1) =
			getn(z, 179, i, k, g, NX, NY, NT+1) +
			getn(rr_right, i, k, g, 0, NX, NT) * h * 1000.f;

		getn(z, 20, i, k, g, NX, NY, NT+1) =
			getn(z, 21, i, k, g, NX, NY, NT+1) +
			getn(rr_left, i, k, g, 0, NX, NT) * h * 1000.f;
	}

	if (i >= 1 && i < (NX - 1)) {
		getn(z, i, 0, k, g, NX, NY, NT+1) =
			getn(z, i, 1, k, g, NX, NY, NT+1);

		getn(z, i, NY-1, k, g, NX, NY, NT+1) =
			getn(z, i, NY-2, k, g, NX, NY, NT+1);

		getn(z, 0, i, k, g, NX, NY, NT+1) =
			getn(z, 1, i, k, g, NX, NY, NT+1);

		getn(z, NX-1, i, k, g, NX, NY, NT+1) =
			getn(z, NX-2, i, k, g, NX, NY, NT+1);
	}

	else if (i == 0) {
		getn(z, 0, 0, k, g, NX, NY, NT+1) =
			(getn(z, 1, 0, k, g, NX, NY, NT+1) +
			 getn(z, 0, 1, k, g, NX, NY, NT+1)) / 2.f;

		getn(z, NX-1, 0, k, g, NX, NY, NT+1) =
			(getn(z, NX-2, 0, k, g, NX, NY, NT+1) +
			 getn(z, NX-1, 1, k, g, NX, NY, NT+1)) / 2.f;

		getn(z, 0, NY-1, k) =
			(getn(z, 1, NY-1, k, g, NX, NY, NT+1) +
			 getn(z, 0, NY-2, k, g, NX, NY, NT+1)) / 2.f;

		getn(z, NX-1, NY-1, k, g, NX, NY, NT+1) =
			(getn(z, NX-2, NY-1, k, g, NX, NY, NT+1) +
			 getn(z, NX-1, NY-2, k, g, NX, NY, NT+1)) / 2.f;
	}
}

__global__ void laplace(const float *u, float *Lu, int g)
{
	// Map from threadIdx / BlockIdx to pixel position

	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;
	int k = threadIdx.z + blockIdx.z * blockDim.z;

	if (i < NX && j < NY && (k + 1) < NT) {

		int j_prev = (j > 0) ? j - 1 : j;
		int j_next = (j < NY - 1) ? j + 1 : j;

		int i_prev = (i > 0) ? i - 1 : i;
		int i_next = (i < NX - 1) ? i + 1 : i;

		getn(Lu, i, j, k+1, g) =
			(getn(u, i, j_prev, k+1, g) +
			 getn(u, i, j_next, k+1, g) +
			 getn(u, i_prev, j, k+1, g) +
			 getn(u, i_next, j, k+1, g) -
			 4.f * getn(u, i, j, k+1, g)) / (h * h);
	}
}

__global__ void laplace_corners(const float *u, float *Lu, int g)
{
	int k = threadIdx.x + blockIdx.x * blockDim.x;

	if ((k + 1) < NT) {
		getn(Lu, 0, 0, k+1, g) =
			(getn(Lu, 1, 0, k+1, g) +
			 getn(Lu, 0, 1, k+1, g)) / 2.f;

		getn(Lu, NX-1, 0, k+1, g) =
			(getn(Lu, NX-2, 0, k+1, g) +
			 getn(Lu, NX-1, 1, k+1, g)) / 2.f;

		getn(Lu, 0, NY-1, k+1, g) =
			(getn(Lu, 1, NY-1, k+1, g) +
			 getn(Lu, 0, NY-2, k+1, g)) / 2.f;

		getn(Lu, NX-1, NY-1, k+1, g) =
			(getn(Lu, NX-2, NY-1, k+1, g) +
			 getn(Lu, NX-1, NY-2, k+1, g)) / 2.f;
	}
}

__global__ void update_differential(float *df, const float *z, const float *Lu, const float *f, int g)
{
	// Map from threadIdx / BlockIdx to pixel position

	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;
	int k = threadIdx.z + blockIdx.z * blockDim.z;

	if(i < NX && j < NY && (k + 1) < NT) {

		atomicAdd(
			&getn(df, i, j),
			getn(z, i, j, k+1, g, NX, NY, NT+1) *
			getn(Lu, i, j, k+1, g) /
			(1.f + getn(f, i, j)));
	}
}

__global__ void print_differential(const float *df)
{
	for (int j = 0; j < NY; ++j) {
		for (int i = 0; i < NX; ++i) {
			printf("%f \t", getn(df, i, j));
		}
		printf("\n");
	}
}

__global__ void update_field(float *f, const float *df, float *f_minus_fo, const float *fo)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;

	if (i < NX && j < NY)
	{
		bool in_sensor_field = (i >= 21) && (i < 180) && (j >= 21) && (j < 180);

		float alpha = in_sensor_field ? 1.f : 0.f;

		getn(f, i, j) += 20000.f * alpha * getn(df, i, j);

		getn(f_minus_fo, i, j) = getn(f, i, j) - getn(fo, i, j);
	}
}

void IO_Files(float *x, float *y, float *fo, float *f)
{
	int i = 0, j = 0;

	// I/O Files

	ofstream x_file, y_file;
	ofstream fo_file;
	ofstream f_file;

	cerr << "writing x to 'dev_x.txt'...\n"
		 << "writing y to 'dev_y.txt'...\n"
		 << "writing f0 to 'dev_f0.txt'...\n"
		 << "writing f to 'dev_f.txt'...\n\n";

	x_file.open("dev_x.txt");
	y_file.open("dev_y.txt");
	fo_file.open("dev_f0.txt");
	f_file.open("dev_f.txt");

	for(i = 0; i < NX; i++) {
		x_file << x[i];
		x_file << "\n";
	}

	for(j = 0; j < NX; j++) {
		y_file << y[j];
		y_file << "\n";
	}

	for(j = 0; j < NY; j++) {
		for(i = 0; i < NX; i++) {
			fo_file << getn(fo, i, j);
			fo_file << " ";
		}

		fo_file << "\n";
	}

	for(j = 0; j < NY; j++) {
		for(i = 0; i < NX; i++) {
			f_file << getn(f, i, j);
			f_file << " ";
		}

		f_file << "\n";
	}

	x_file.close();
	y_file.close();
	fo_file.close();
	f_file.close();
}

float norm(float *A, int nx, int ny)
{
	float sum = 0;

	for (int j = 0; j < ny; ++j)
		for (int i = 0; i < nx; ++i)
			sum += getn(A, i, j) * getn(A, i, j);

	return sqrtf(sum);
}
